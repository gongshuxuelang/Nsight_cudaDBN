#include "hip/hip_runtime.h"
#include "head.h"


//接收的数据为32*7681,32行互不相干。开32个线程来处理这个矩阵。d_data是数据row是行，line是列，DBN_N是分解层数
__global__ void GPU0(double* d_a,double* max,int row,int line,int dbn,int dbn_n)
{
	int idx = threadIdx.x;
	switch(idx)
	{
	case 0:
		dec(d_a,max,row,line,dbn,idx,dbn_n);
		__syncthreads();
		break;
	case 1:

		__syncthreads();
		break;
	case 2:

		__syncthreads();
		break;
	case 3:

		__syncthreads();
		break;
	}
}

__global__ void GPU1(double* d_a,int row,int line,int dbn,int dbn_n)
{


}

__device__ void printMax(double* max,int dbn)
{
	printf("GPU打印\n");
    for(int i = 0; i < 4; ++i)
    {
        for(int j = 0; j < 2 * dbn; ++j)
        {
        	printf("%f\t",max[i * 2 * dbn + j]);
        }
        printf("\n");
    }
}

__device__ void printData(double* buffer,int row,int line)
{
    for(int i = 0; i < row; ++i)
    {
        for(int j = 0; j < line; ++j)
        {
        	printf("%d\t",buffer[i * line + j]);
        }
        printf("\n");
    }
}
__device__ void decTransfromEx(double* d_data,double* dVectorUpSam,int line,int idx,int dbn,int decExLen,double* dVectorEx,int DBN_N)
{
	if(DBN_N == 0)
	{
		for(int i = 0; i < 2 * dbn - 1; ++i)
		{
			dVectorEx[i] 					  = d_data[idx * line + 2 * dbn -2 - i];
			dVectorEx[2 * dbn + line - 1 + i] = d_data[idx * line + line -1 - i];
		}
		for(int i = 0; i < line; ++i)
		{
			dVectorEx[2 * dbn - 1 + i] = d_data[idx * line + i];
		}
	}else{
		for(int i = 0; i < 2 * dbn - 1; ++i)
		{
			dVectorEx[i] 					  = dVectorUpSam[idx * line + 2 * dbn -2 - i];
			dVectorEx[2 * dbn + line - 1 + i] = dVectorUpSam[idx * line + line -1 - i];
		}
		for(int i = 0; i < line; ++i)
		{
			dVectorEx[2 * dbn - 1 + i] = dVectorUpSam[idx * line + i];
		}
	}

	//打印延拓结果
	for(int i = 0; i < decExLen; ++i)
	{
		printf("dVectorEx = %f ",dVectorEx[i]);
	}
	printf("\n");
	return ;
}
__global__ void dataCON(double* dVectorEx,double* sdecMaxL,double* sdecMaxH,int dbn,int decCONLen,double* dVectorCON,int idx,int decExLen,int DBN_N)
{
	double tempL = 0;
	double tempH = 0;
	int iidx = threadIdx.x;
	for(int i = 0; i < 2* dbn; ++i)
	{
		tempL += dVectorEx[iidx + i] * sdecMaxL[i];
		tempH += dVectorEx[iidx + i] * sdecMaxH[i];
	}
	dVectorCON[iidx] = tempL;
	dVectorCON[iidx + decCONLen] = tempH;
}
__device__ void decTransfromCON(double* dVectorEx,double* max,int line,int idx,int decCONLen,int dbn,double* dVectorCON,int decExLen,int DBN_N)
{
	double* sdecMaxL= new double[2 * dbn];
	double* sdecMaxH= new double[2 * dbn];

	for(int i = 0; i < 2 * dbn; ++i)
	{
		sdecMaxL[i] = max[0 * 2 * dbn + i];
		sdecMaxH[i] = max[1 * 2 * dbn + i];
		printf("sdecMaxL = %f\t",sdecMaxL[i]);
		printf("\n");
		printf("sdecMaxH = %f\t",sdecMaxH[i]);
		printf("\n");
	}
	dataCON<<<1,decCONLen>>>(dVectorEx,sdecMaxL,sdecMaxH,dbn,decCONLen,dVectorCON,idx,decExLen,DBN_N);
	__syncthreads();
	hipFree(sdecMaxL);
	hipFree(sdecMaxH);
}
__global__ void decUpSam(double* dVectorCON,double*dVectorUpSam,int decCONLen,int decUpSamLen,int DBN_N)
{
	int iUdx = threadIdx.x;
	dVectorUpSam[iUdx] 				 = dVectorCON[2 * iUdx + 1];
	dVectorUpSam[decUpSamLen + iUdx] = dVectorCON[decCONLen + 2 * iUdx + 1];
}
__device__ void dec(double* d_a,double* max,int row,int line,int dbn,int idx,int dbn_n)
{
	for(int DBN_N = 0; DBN_N < dbn_n; ++DBN_N)
	{
		int decExLen    = line + 4 * dbn - 2;//对称延拓长度
		int decCONLen   = line + 2 * dbn - 1;//分解卷积长度
		int decUpSamLen = (line + 2* dbn -1) / 2;//上采样长度

		double* dVectorEx 	  = new double[power(2,DBN_N) * decExLen];
		double* dVectorCON    = new double[power(2,DBN_N + 1) * decCONLen];		//  卷积和内存
		double* dVectorUpSam;													//  上采样内存
//		if(DBN_N == dbn_n - 1)
//		{
//			d_line = decUpSamLen;
//
//		}
		decTransfromEx(d_a,dVectorUpSam,line,idx,dbn,decExLen,dVectorEx,DBN_N);//延拓

		for(int i = 0; i < decExLen;++i)
		{
			printf("case:dVectorEx = %f\t",dVectorEx[i]);
		}
		printf("\n");
		decTransfromCON(dVectorEx,max,line,idx,decCONLen,dbn,dVectorCON,decExLen,DBN_N);//卷积
		__syncthreads();
		hipFree(dVectorEx);
		printf("CON=\n");
		printf("decCONLen = %d\n",decCONLen);
		for(int i = 0; i < decCONLen;++i)
		{
			printf("dVectorCONL[%d] = %f\n  ",i,dVectorCON[i]);
			printf("dVectorCONH[%d] = %f\n  ",i,dVectorCON[i + decCONLen]);
		}
		dVectorUpSam = new double[power(2,DBN_N + 1) * decUpSamLen];
		decUpSam<<<1,decUpSamLen>>>(dVectorCON,dVectorUpSam,decCONLen,decUpSamLen,DBN_N);
		hipFree(dVectorCON);
		printf("UpSam=\n");
		printf("decUpSamLen = %d\n",decUpSamLen);
		for(int i = 0; i < decUpSamLen;++i)
		{
			printf("dVectorUpSamL[%d] = %f\n  ",i,dVectorUpSam[i]);
			printf("dVectorUpSamH[%d] = %f\n  ",i,dVectorUpSam[i + decUpSamLen]);
		}
		for(int i = 0; i < 2 * decUpSamLen;++i)
		{
			printf("dVectorUpSam[%d] = %f\n  ",i,dVectorUpSam[i]);
		}
		//hipFree(dVectorUpSam);
		//更新系数,进行下一层分解
		line = decUpSamLen;
	}
}
__device__ int power(int base, int exponent)
{
	int result = 1;
	if(exponent == 0)
	{
		return result;
	}
	for (int i = 0; i < exponent; ++i)
	{
		result *= base;
	}

	return result;
}
