#include "hip/hip_runtime.h"
#include "head.h"


//接收的数据为32*7681,32行互不相干。开32个线程来处理这个矩阵。d_data是数据row是行，line是列，DBN_N是分解层数
__global__ void GPU0(double* d_a,double* max,int row,int line,int dbn,int dbn_n)
{
	//初始化重要的系数系数
	printf("GPU\n");
	int tid = threadIdx.x;
	int decExLen  = line + 4 * dbn - 2;//对称延拓长度
	int decCONLen = line + 2 * dbn - 1;//f分解卷积长度
	double* d_data = d_a;
	//构造产量内存
//	__shared__ double* sdecLo;
//	__shared__ double* sdecHo;
//	__shared__ double* ssamLo;
//	__shared__ double* ssamHo;
//	for(int i = 0; i < 2 * dbn; ++i)
//	{
//		sdecLo[i] = max[0 * dbn + i];
//		sdecHo[i] = max[1 * dbn + i];
//		ssamLo[i] = max[2 * dbn + i];
//		ssamHo[i] = max[3 * dbn + i];
//	}
	switch(tid)
	{
	case 0:
		printf("EX\n");
		double* dVectorEx = new double[decExLen];
		dVectorEx = decTransfromEx(d_data,line,tid,dbn,decExLen);//延拓
		for(int i = 0; i < decExLen;++i)
		{
			printf("%d\t",dVectorEx[i]);
		}
		printf("\n");
		decTransfromCON(dVectorEx,max,line,tid,decCONLen,dbn);//卷积

		break;
	case 1:
		break;
	case 2:
		break;
	case 3:
		break;
	}
}

__global__ void GPU1(double* d_a,int row,int line,int dbn,int dbn_n)
{


}

__device__ void printMax(double* max,int dbn)
{
	printf("GPU打印\n");
    for(int i = 0; i < 4; ++i)
    {
        for(int j = 0; j < 2 * dbn; ++j)
        {
        	printf("%f\t",max[i * 2 * dbn + j]);
        }
        printf("\n");
    }
}

__device__ void printData(double* buffer,int row,int line)
{
    for(int i = 0; i < row; ++i)
    {
        for(int j = 0; j < line; ++j)
        {
        	printf("%d\t",buffer[i * line + j]);
        }
        printf("\n");
    }
}
__device__ double* decTransfromEx(double* d_data,int line,int tid,int dbn,int decExLen)
{
	double* dVectorEx = new double[decExLen];
	for(int i = 0; i < 2 * dbn - 1; ++i)
	{
		dVectorEx[i] = d_data[tid * line + 2 * dbn -2 - i];
		dVectorEx[2 * dbn + line - 1 + i] = d_data[tid * line + line -1 - i];
	}
	for(int i = 0; i < line; ++i)
	{
		dVectorEx[2 * dbn - 1 + i] = d_data[tid * line + i];
	}
	return dVectorEx;
}
__global__ void dataCON(double* dVectorEx,double* sdecMaxL,double* sdecMaxH,int dbn,int decCONLen,double* dVectorCON)
{

	int tidx = threadIdx.x;
	int bidx = blockIdx.x;
	double temp = 0;
	int i= 0,j = 0,ll = 0;
	for(j = bidx * blockDim.x + tidx;j < decCONLen; j+=gridDim.x * blockDim.x)
	{
		temp = 0;
		for(i = 0; i < decCONLen; ++i)
		{
			ll = j + 1;
			temp += dVectorEx[ll] *  sdecMaxL[i];
		}
		dVectorCON[j] = temp;
	}
}
__device__ double* decTransfromCON(double* dVectorEx,double* max,int line,int tid,int decCONLen,int dbn)
{
	double* sdecMaxL= new double[2 * dbn];
	double* sdecMaxH= new double[2 * dbn];
	double* dVectorCON = new double[decCONLen];
	for(int i = 0; i < 2 * dbn; ++i)
	{
		sdecMaxL[i] = max[0 * 2 * dbn + i];
		sdecMaxH[i] = max[1 * 2 * dbn + i];
		printf("sdecMaxL = %f\t",sdecMaxL[i]);
		printf("\n");
		printf("sdecMaxH = %f\t",sdecMaxH[i]);
		printf("\n");
	}
	dataCON<<<1,4>>>(dVectorEx,sdecMaxL,sdecMaxH,dbn,decCONLen,dVectorCON);

	for(int i = 0; i < 4;++i)
	{
		for(int j = 0; j < decCONLen; ++j)
		{
			printf("%f\t",dVectorCON[i * decCONLen + j]);
		}
		printf("\n");
	}
	return dVectorCON;
}
