#include "hip/hip_runtime.h"
#include "head.h"
/*
 * 说明,GPU分出一个文件的向量,例如32个行向量,需要分出32个线程.
 * 进入GPU中每一个线程又需要干不同的任务,一个行向量要进行DBN_N次分解和重构
 * 每一次分解要比上一次的行向量多2倍,总体上就是2的DBN_N次幂的关系.
 * 我需要的做的就是理清楚一个行向量做出2维数据,用并行算法写出代码.避免串行代码的问题,既然使用了GPU那就不要浪费GPU的计算性能.
 * */


__shared__ double* sdecMaxL;
__shared__ double* sdecMaxH;

__device__ int T_power(int n)
{
	int m = 1;
	m = m << n;
	return m;
}
__device__ int power(int base, int exponent)
{
	int result = 1;
	if(exponent == 0)
	{
		return result;
	}
	for (int i = 0; i < exponent; ++i)
	{
		result *= base;
	}

	return result;
}

__device__ void dec(double* d_a,double* max,int row,int line,int dbn,int idx,int dbn_n,double* refdata)
{
	for(int DBN_N = 0; DBN_N < dbn_n; ++DBN_N)
	{
//		int decExLen    = line + 4 * dbn - 2;//对称延拓长度
//		int decCONLen   = line + 2 * dbn - 1;//分解卷积长度
//		int decUpSamLen = (line + 2* dbn -1) / 2;//上采样长度////
//		decTransfromEx(d_a,dVectorUpSam,line,idx,dbn,decExLen,dVectorEx,DBN_N);//延拓
//		__syncthreads();
//		printf("decExLen = %d\n",decExLen);

//		for(int i = 0;i < row * T_power(DBN_N) * decExLen; ++i)
//		{
//			printf("dVectorEx[%d] = %f\n",i,*(dVectorEx + i));
//		}
//		decTransfromCON<<<1,T_power(DBN_N)>>>(dVectorEx,sdecMaxL,sdecMaxH,decCONLen,dbn,dVectorCON,decExLen,DBN_N);//卷积
//		__syncthreads();

//		if(DBN_N != 0)
//		{
//			delete[] dVectorUpSam;
//		}
		//只能在这里申请内存,在开头申请内存第二次循环数据为空了.
//
//		dVectorUpSam = new double[T_power(DBN_N + 1) * decUpSamLen]();
//		dVectorUpSam = new double[10]();
//		delete[] dVectorEx;
//		delete[] dVectorCON;
//		delete[] dVectorUpSam;//
//		decTransfromUpSam<<<1,T_power(DBN_N + 1)>>>(dVectorCON,dVectorUpSam,decCONLen,decUpSamLen,DBN_N);
//		__syncthreads();
		//更新系数,进行下一层分解
//		line = decUpSamLen;
//		if(dbn_n - 1 == DBN_N)//重构信号长度
//		{
//			memcpy(refdata + line * T_power(dbn_n) * idx,dVectorUpSam,sizeof(double) * line * T_power(dbn_n));
//
//			//delete[] dVectorUpSam;
//		}

	}

}

__device__ void decSignal(double* DeviceDecDataFinsh,double* dVectorEx,double* dVectorCON,double* d_a,double* max,int row,int line,int dbn,int dbn_n,int idx)
{
	sdecMaxL = new double[2 * dbn]();
	sdecMaxH = new double[2 * dbn]();
	for(int i = 0; i < 2 * dbn; ++i)
	{
		sdecMaxL[i] = max[0 * 2 * dbn + i];
		sdecMaxH[i] = max[1 * 2 * dbn + i];
	}
	for(int DBN_N = 0; DBN_N < 1; ++DBN_N)
	{
		int decExLen    = line + 4 * dbn - 2;//对称延拓长度
		int decCONLen   = line + 2 * dbn - 1;//分解卷积长度
		int decUpSamLen = (line + 2* dbn -1) / 2;//上采样长度
		decTransfromEx(d_a,DeviceDecDataFinsh,line,idx,dbn,decExLen,dVectorEx,DBN_N);//延拓
		__syncthreads();
		for(int i = 0;i < row * T_power(DBN_N) * decExLen; ++i)
		{
			printf("dVectorEx[%d] = %f\n",i,*(dVectorEx + i));
		}

		dim3 dimBlock;
		if(0 == decCONLen / 1000)
		{
			dimBlock = (1,decCONLen);
		}else{
			int girdconIdx  = 0;
			int blockconIdx = 0;
			girdconIdx  = decCONLen / 1000 + 1;
			blockconIdx = 1000;
			dimBlock = (girdconIdx,blockconIdx);
		}
		decTransfromCON<<<T_power(DBN_N),dimBlock>>>(dVectorEx,sdecMaxL,sdecMaxH,decCONLen,dbn,dVectorCON,decExLen,DBN_N,idx);//卷积
		__syncthreads();
		printf("dVectorCON[%d] = %f\n",0,*(dVectorCON));
		printf("dVectorCON = %d\n",dVectorCON);
		for(int i = 0;i < T_power(DBN_N+1) * decCONLen; ++i)
		{
			printf("i = %d\n",i);
			printf("dVectorCON = %d\n",dVectorCON+i);
			printf("dVectorCON[%d] = %f\n",idx * T_power(DBN_N+1) * decCONLen + i,*(dVectorCON + idx * T_power(DBN_N+1) * decCONLen + i));
		}
	}
}
//接收的数据为32*7681,32行互不相干。开32个线程来处理这个矩阵。d_data是数据row是行，line是列，DBN_N是分解层数
__global__ void GPU0(double* DeviceDecDataFinsh,double* dVectorEx,double* dVectorCON,double* d_a,double* max,int row,int line,int dbn,int dbn_n,int finshline)
{
	int idx = threadIdx.x;
	decSignal(DeviceDecDataFinsh,dVectorEx,dVectorCON,d_a,max,row,line,dbn,dbn_n,idx);
}

__device__ void ref(double* refdata,double* max,int refline,int dbn,int dbn_n,int Begin,int End)
{
	double* refMaxL= new double[2 * dbn]();
	double* refMaxH= new double[2 * dbn]();
	for(int i = 0; i < 2 * dbn; ++i)
	{
		refMaxL[i] = max[2 * 2 * dbn + i];
		refMaxH[i] = max[3 * 2 * dbn + i];
		printf("refMaxL[%d] = %f\n",i,refMaxL[i]);
		printf("refMaxH[%d] = %f\n",i,refMaxH[i]);
	}
	printf("refline = %d\n",refline);
	for(int i = 0; i < 24; ++i)
	{
		printf("refdata[%d] = %f\n",i,refdata[i]);
	}
	for(int DBN_N = 0; DBN_N < 1; ++DBN_N)
	{
		printf("refline = %d\n",refline);
		for(int i = 0; i < 24; ++i)
		{
			printf("refdata[%d] = %f\n",i,refdata[i]);
		}

		int refDsamLen   = 2 * refline + 1;				//重构下采样长度
		int refExLen     = 2 * refline + 4 * dbn - 1;	//重构对称延拓长度
		int refConLen    = 2 * refline + 2 * dbn;		//重构卷积长度
		int refSingalLen = 2 * refline - 2 * dbn + 2;	//重构信号长度

		double* refTransfromDsam = new double[(End - Begin + 1) * refDsamLen]();
		double* refTransfromEx   = new double[(End - Begin + 1) * refExLen]();
		double* refTransfromCon  = new double[(End / 2 - Begin / 2 + 1) * refConLen]();
		double* refTransfromSignal;


		refSignalDSam<<<1,End - Begin + 1>>>(refdata,refTransfromDsam,refTransfromSignal,refDsamLen,Begin,DBN_N,refline);
		__syncthreads();
		printf("refDsamLen = %d\n",refDsamLen);
		for(int i = 0; i < (End - Begin + 1) * refDsamLen; ++i)
		{
			printf("refTransfromDsam[%d] = %f\n",i,refTransfromDsam[i]);
		}
		printf("refExLen = %d\n",refExLen);
		refSignalEx<<<1,End - Begin + 1>>>(refTransfromDsam,refTransfromEx,dbn,refDsamLen,refExLen);
		__syncthreads();
		for(int i = 0; i < (End - Begin + 1) * refExLen; ++i)
		{
			printf("refTransfromEx[%d] = %f\n",i,refTransfromEx[i]);
		}
		delete[] refTransfromDsam;
		//卷积区分高频低频卷积
		//测试卷积效果

		printf("Begin = %d\n",Begin);
		printf("End = %d\n",End);
		if(Begin % 2 == 0)
		{
			if(End % 2 == 0)//第一种情况 Begin和End都是偶数
			{
				printf("执行第一种情况\n");
				refSignalCon<<<1,End / 2 - Begin / 2>>>(refTransfromEx,refTransfromCon,refMaxL,refMaxH,refConLen,refExLen,dbn,Begin,End,0);
				refConEnd<<<1,refConLen>>>(refTransfromEx,refTransfromCon,refMaxL,refConLen,refExLen,dbn,Begin,End);//计算End数据
			}else{//第二种情况Begin是偶数,End是奇数
				printf("执行第二种情况\n");
				refSignalCon<<<1,End / 2 - Begin / 2 + 1>>>(refTransfromEx,refTransfromCon,refMaxL,refMaxH,refConLen,refExLen,dbn,Begin,End,0);
			}
		}else{
			if(End % 2 == 0)//第三种情况Begin是奇数,End是偶数
			{
				printf("执行第三种情况\n");
				refConBegin<<<1,refConLen>>>(refTransfromEx,refTransfromCon,refMaxH,dbn);//计算Begin数据
				refSignalCon<<<1,End / 2 - Begin / 2 - 1>>>(refTransfromEx,refTransfromCon,refMaxL,refMaxH,refConLen,refExLen,dbn,Begin,End,1);
				refConEnd<<<1,refConLen>>>(refTransfromEx,refTransfromCon,refMaxL,refConLen,refExLen,dbn,Begin,End);//计算End数据
			}else{//第四种情况,Begin和End都是奇数
				printf("执行第四种情况\n");
				refConBegin<<<1,refConLen>>>(refTransfromEx,refTransfromCon,refMaxH,dbn);//计算Begin数据
				refSignalCon<<<1,End / 2 - Begin / 2>>>(refTransfromEx,refTransfromCon,refMaxL,refMaxH,refConLen,refExLen,dbn,Begin,End,1);
			}
		}
		delete[] refTransfromEx;
		//打印卷积结果
		printf("refTransfromCon = \n");
		printf("refConLen = %d\n",refConLen);
		for(int i = 0; i < (End / 2 - Begin / 2 + 1) * refConLen; ++i)
		{
			printf("refTransfromCon[%d] = %f\n",i,refTransfromCon[i]);
		}
		//信号选取
		refTransfromSignal = new double[(End / 2 - Begin / 2 + 1) * refSingalLen]();
		refSignal<<<1,End / 2 - Begin / 2 + 1>>>(refTransfromCon,refTransfromSignal,refConLen,refSingalLen,dbn);
		delete[] refTransfromCon;

		printf("refTransfromSignal = \n");
		printf("refSingalLen = %d\n",refSingalLen);
		for(int i = 0; i < (End / 2 - Begin / 2 + 1) * refSingalLen; ++i)
		{
			printf("refTransfromSignal[%d] = %f\n",i,refTransfromSignal[i]);
		}
		Begin = (int)Begin / 2;
		End   = (int)End / 2;
		printf("Begin = %d\n",Begin);
		printf("End = %d\n",End);
	}
	delete[] refMaxL;
	delete[] refMaxH;
}

__global__ void refSignal(double* refTransfromCon,double*refTransfromSignal,int refConLen,int refSingalLen,int dbn)
{
	int refSignalIdx = threadIdx.x;
	refSig<<<1,refSingalLen>>>(refTransfromCon,refTransfromSignal,refConLen,refSingalLen,dbn,refSignalIdx);

}

__global__ void refSig(double* refTransfromCon,double*refTransfromSignal,int refConLen,int refSingalLen,int dbn,int refSignalIdx)
{
	int refSinidx = threadIdx.x;
	refTransfromSignal[refSignalIdx * refSingalLen + refSinidx] = refTransfromCon[refSignalIdx * refConLen + 2 * dbn - 2 + refSinidx];
}

//flag是为了保证Begin和之后的不会重复
__global__ void refConBegin(double* refTransfromEx,double* refTransfromCon,double* refMaxH,int dbn)
{
	int refConidx = threadIdx.x;
	for(int i = 0; i < 2* dbn; ++i)
	{
		refTransfromCon[refConidx] += refTransfromEx[refConidx + i] * refMaxH[i];
	}
}

__global__ void refConEnd(double* refTransfromEx,double* refTransfromCon,double* refMaxL,int refConLen,int refExLen,int dbn,int Begin,int End)
{
	int refConidx = threadIdx.x;
	for(int i = 0; i < 2* dbn; ++i)
	{
		refTransfromCon[(End / 2 - Begin / 2) * refConLen + refConidx] += refTransfromEx[(End - Begin) * refExLen + refConidx + i] * refMaxL[i];
	}
}

__global__ void refCon(double* refTransfromEx,double* refTransfromCon,double* refMaxL,double* refMaxH,int refConLen,int refExLen,int dbn,int refConBlockIdx,int flag)
{
	int refConidx = threadIdx.x;

	for(int i = 0; i < 2* dbn; ++i)
	{
		refTransfromCon[(refConBlockIdx + flag) * refConLen + refConidx] += refTransfromEx[2 * refConBlockIdx * refExLen + flag  * refExLen + refConidx + i] * refMaxL[i]
		                                                        + refTransfromEx[(2 * refConBlockIdx + 1) * refExLen  + flag * refExLen + refConidx + i] * refMaxH[i];
	}
}

__global__ void refSignalCon(double* refTransfromEx,double* refTransfromCon,double* refMaxL,double* refMaxH,int refConLen,int refExLen,int dbn,int Begin,int End,int flag)
{
	int refConBlockIdx = threadIdx.x;

	refCon<<<1,refConLen>>>(refTransfromEx,refTransfromCon,refMaxL,refMaxH,refConLen,refExLen,dbn,refConBlockIdx,flag);
}

__device__ void refEx(double* refTransfromDsam,double* refTransfromEx,int dbn,int refDsamLen,int refExLen,int refExidx)
{
	for(int i = 0; i < 2 * dbn - 1; ++i)
	{
		refTransfromEx[refExidx * refExLen + i] 				       = refTransfromDsam[refExidx * refDsamLen + 2 * dbn -2 - i];
		refTransfromEx[refExidx * refExLen + refDsamLen + 2 * dbn -1 + i] = refTransfromDsam[refExidx * refDsamLen + refDsamLen -1 - i];
	}
	for(int i = 0; i < refDsamLen; ++i)
	{
		refTransfromEx[refExidx * refExLen + 2 * dbn - 1 + i] = refTransfromDsam[refExidx * refDsamLen + i];
	}
}

__global__ void refSignalEx(double* refTransfromDsam,double* refTransfromEx,int dbn,int refDsamLen,int refExLen)
{
	int refExidx = threadIdx.x;
	refEx(refTransfromDsam,refTransfromEx,dbn,refDsamLen,refExLen,refExidx);
}

__global__ void refDSam(double* refdata,double* refTransfromDsam,double* refTransfromSignal,int refDsamLen,int Begin,int DBN_N,int refline,int refDsamidx)
{
	int refDidx = threadIdx.x;
	if(DBN_N == 0)
	{
		refTransfromDsam[refDsamidx * refDsamLen + 2 * refDidx + 1] = refdata[(refDsamidx + Begin) * refline + refDidx];
	}else{
		refTransfromDsam[refDsamidx * refDsamLen + 2 * refDidx + 1] = refTransfromSignal[refDsamidx * refDsamLen + refDidx];
	}
}

__global__ void refSignalDSam(double* refdata,double* refTransfromDsam,double* refTransfromSignal,int refDsamLen,int Begin,int DBN_N,int refline)
{
	int refDsamidx = threadIdx.x;
	refDSam<<<1,refline>>>(refdata,refTransfromDsam,refTransfromSignal,refDsamLen,Begin,DBN_N,refline,refDsamidx);
	__syncthreads();
}

/*
 * 选择波形一共四种波形,Alpha,Beta,Delta,Theta,指定开四个线程,每个线程代表一个波形的重构,
 * 0线程代表Alpha,1线程代表Beta,2线程代表Delta,3线程代表Theta
 *
 * */
__global__ void refChooseSignal(double* refdata,double* max,int refline,int dbn,int dbn_n)
{
	int refchSigidx = threadIdx.x;

//	switch(refchSigidx)
//	{
//	case 0:
//		/*
//		 * Alpha波的起始频率为7.81Hz终止信号为13.28Hz
//		 *
//		 * */
//		printf("选择Alpha波.\n");
////		int AlphaBegin = static_cast<int>(floor(7.81 * power(2,dbn_n) / 64));
////		int AlphaEnd   = static_cast<int>(floor(13.28 * power(2,dbn_n) / 64));
//		int AlphaBegin = 0;
//		int AlphaEnd = 1;
//		printf("AlphaBegin = %d\n",AlphaBegin);
//		printf("AlphaEnd = %d\n",AlphaEnd);
//		ref(refdata,max,refline,dbn,dbn_n,AlphaBegin,AlphaEnd);
//
//
//		break;
//	case 1:
//		/*
//		 * Beta波的起始频率为13.28Hz终止信号为30.47Hz
//		 *
//		 * */
//		printf("选择Beta波.\n");
//		int BetaBegin = static_cast<int>(floor(13.28 * power(2,dbn_n) / 64));
//		int BetaEnd   = static_cast<int>(floor(30.47 * power(2,dbn_n) / 64));
//		printf("BetaBegin = %d\n",BetaBegin);
//		printf("BetaEnd = %d\n",BetaEnd);
//		break;
//	case 2:
//		/*
//		 * Delta波的起始频率为0.78Hz终止信号为3.91Hz
//		 *
//		 * */
//		printf("选择Delta波.\n");
//		int DeltaBegin = static_cast<int>(floor(0.78 * power(2,dbn_n) / 64));
//		int DeltaEnd   = static_cast<int>(floor(3.91 * power(2,dbn_n) / 64));
//		printf("DeltaBegin = %d\n",DeltaBegin);
//		printf("DeltaEnd = %d\n",DeltaEnd);
//		break;
//	case 3:
//		/*
//		 *Theta波的起始频率为3.91Hz终止信号为7.81Hz
//		 *
//		 * */
//		printf("选择Theta波.\n");
//		int ThetaBegin = static_cast<int>(floor(3.91 * power(2,dbn_n) / 64));
//		int ThetaEnd   = static_cast<int>(floor(7.81 * power(2,dbn_n) / 64));
//		printf("ThetaBegin = %d\n",ThetaBegin);
//		printf("ThetaEnd = %d\n",ThetaEnd);
//		break;
//	}


}

__global__ void GPU1(double* d_a,int row,int line,int dbn,int dbn_n)
{


}

__device__ void printMax(double* max,int dbn)
{
	printf("GPU打印\n");
    for(int i = 0; i < 4; ++i)
    {
        for(int j = 0; j < 2 * dbn; ++j)
        {
        	printf("%f\t",max[i * 2 * dbn + j]);
        }
        printf("\n");
    }
}

__device__ void printData(double* buffer,int row,int line)
{
    for(int i = 0; i < row; ++i)
    {
        for(int j = 0; j < line; ++j)
        {
        	printf("%d\t",buffer[i * line + j]);
        }
        printf("\n");
    }
}
// 这个函数用于开线程延拓线程开一组的就可以其他组的用line来区分
__global__ void decEx(double* dVectorUpSam,int line,int dbn,double*& dVectorEx)
{
	int decExidx = threadIdx.x;//这个线程号最多只有一组的线程号
	for(int i = 0; i < 2 * dbn - 1; ++i)
	{
		dVectorEx[decExidx  * (line + 4 * dbn - 2) + i] 					 = dVectorUpSam[decExidx * line + 2 * dbn -2 - i];
		dVectorEx[decExidx  * (line + 4 * dbn - 2) + 2 * dbn + line - 1 + i] = dVectorUpSam[decExidx * line + line -1 - i];
	}
	for(int i = 0; i < line; ++i)
	{
		dVectorEx[decExidx  * (line + 4 * dbn - 2) + 2 * dbn - 1 + i] = dVectorUpSam[decExidx  * line + i];
	}
}

__device__ void decTransfromEx(double* d_data,double* dVectorUpSam,int line,int idx,int dbn,int decExLen,double*& dVectorEx,int DBN_N)
{
	if(DBN_N == 0)
	{
		for(int i = 0; i < 2 * dbn - 1; ++i)
		{
			dVectorEx[idx * T_power(DBN_N) * decExLen + i] = d_data[idx * line + 2 * dbn -2 - i];
			dVectorEx[idx * T_power(DBN_N) * decExLen + 2 * dbn + line - 1 + i] = d_data[idx * line + line -1 - i];
		}
		for(int i = 0; i < line; ++i)
		{
			dVectorEx[idx * T_power(DBN_N) * decExLen + 2 * dbn - 1 + i] = d_data[idx * line + i];
		}
	}else{
		decEx<<<1,T_power(DBN_N)>>>(dVectorUpSam,line,dbn,dVectorEx);
		__syncthreads();
	}
}

__global__ void decTransfromCON(double* dVectorEx,double* sdecMaxL,double*sdecMaxH,int decCONLen,int dbn,double* dVectorCON,int decExLen,int DBN_N,int idx)
{
	int decCONidx = threadIdx.y * blockDim.x + threadIdx.x;
	int decgridIdx = blockIdx.x;

	for(int i = 0; i < 2 * dbn; ++i)
	{
		dVectorCON[idx * T_power(DBN_N + 1) * decCONLen + 2 * decgridIdx * decCONLen + decCONidx]       += dVectorEx[idx * T_power(DBN_N) * decExLen + decgridIdx * decExLen + decCONidx + i] * sdecMaxL[i];
		dVectorCON[idx * T_power(DBN_N + 1) * decCONLen + (2 * decgridIdx + 1) * decCONLen + decCONidx] += dVectorEx[idx * T_power(DBN_N) * decExLen + decgridIdx * decExLen + decCONidx + i] * sdecMaxH[i];
	}
}

__global__ void decUpSam(double* dVectorCON,double* dVectorUpSam,int decCONLen,int decUpSamLen,int DBN_N,int decUpidx)
{
	int iUpdx = blockIdx.x * blockDim.x + threadIdx.x;
	dVectorUpSam[decUpSamLen * decUpidx + iUpdx] = dVectorCON[decCONLen * decUpidx + 2 * iUpdx + 1];
}

__global__ void decTransfromUpSam(double* dVectorCON,double* dVectorUpSam,int decCONLen,int decUpSamLen,int DBN_N)
{
	int decUpidx = threadIdx.x;
	int gridUpsamIdx  = 0;
	int blockUpsamIdx = 0;

	if(decUpSamLen >= 1000)
	{
		gridUpsamIdx  = decUpSamLen / 1000 + 1;
		blockUpsamIdx = 1000;
	}else{
		gridUpsamIdx  = 1;
		blockUpsamIdx = decUpSamLen;
	}
	decUpSam<<<gridUpsamIdx,blockUpsamIdx>>>(dVectorCON,dVectorUpSam,decCONLen,decUpSamLen,DBN_N,decUpidx);
	__syncthreads();
}

