#include "hip/hip_runtime.h"
#include "head.h"


//接收的数据为32*7681,32行互不相干。开32个线程来处理这个矩阵。d_data是数据row是行，line是列，DBN_N是分解层数
__global__ void GPU0(double* d_a,int row,int line,int dbn,int dbn_n)
{
	//printf("GPU操作\n");
	double* d_data = d_a;
	int tid = threadIdx.x;
	switch(tid)
	{
	case 0:
		decTransfromEx(d_data,line,tid,dbn);


		break;
	case 1:
		decTransfromEx(d_data,line,tid,dbn);
		break;
	case 2:
		decTransfromEx(d_data,line,tid,dbn);
		break;
	case 3:
		decTransfromEx(d_data,line,tid,dbn);
		break;
	}
}

__global__ void GPU1(double* d_a,int row,int line,int dbn,int dbn_n)
{


}

__device__ void decTransfromEx(double* d_data,int line,int tid,int dbn)
{
	int decExLen = line + 4 * dbn - 2;
	double* dVectorEx = new double[decExLen];
	for(int i = 0; i < 2 * dbn - 1; ++i)
	{
		dVectorEx[i] = d_data[tid * line + 2 * dbn -2 - i];
		dVectorEx[2 * dbn + line - 1 + i] = d_data[tid * line + line -1 - i];
	}
	for(int i = 0; i < line; ++i)
	{
		dVectorEx[2 * dbn - 1 + i] = d_data[tid * line + i];
	}
	for(int i = 0; i < decExLen; ++i)
	{
		printf("%f\t",dVectorEx[i]);
	}
}

