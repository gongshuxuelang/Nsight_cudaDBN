#include "hip/hip_runtime.h"
#include "head.h"
/*
 * 说明,GPU分出一个文件的向量,例如32个行向量,需要分出32个线程.
 * 进入GPU中每一个线程又需要干不同的任务,一个行向量要进行DBN_N次分解和重构
 * 每一次分解要比上一次的行向量多2倍,总体上就是2的DBN_N次幂的关系.
 * 我需要的做的就是理清楚一个行向量做出2维数据,用并行算法写出代码.避免串行代码的问题,既然使用了GPU那就不要浪费GPU的计算性能.
 * */
__device__ int power(int base, int exponent)
{
	int result = 1;
	if(exponent == 0)
	{
		return result;
	}
	for (int i = 0; i < exponent; ++i)
	{
		result *= base;
	}

	return result;
}
//接收的数据为32*7681,32行互不相干。开32个线程来处理这个矩阵。d_data是数据row是行，line是列，DBN_N是分解层数
__global__ void GPU0(double* DeviceDecDataFinsh,double* d_a,double* max,int row,int line,int dbn,int dbn_n,int finshline)
{
	int idx = threadIdx.x;
	printf("idx = %d\n",idx);
	double* refdata = new double[row * power(2,dbn_n) * finshline]();
	switch(idx)
	{
	case 0:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		for(int i = 0; i < finshline; ++i)
		{
			printf("refdata[%d] =  %0.16f\n",i,refdata[i]);
			printf("DeviceDecDataFinsh[%d] =  %0.16f\n",i,DeviceDecDataFinsh[i]);
		}
//		/*重构部分*/
//		printf("refline = %d\n",refline);
//		printf("power(2,dbn_n) * refline = %d\n",power(2,dbn_n) * refline);
//		for(int i = 0; i < power(2,dbn_n) * refline;++i)
//		{
//			printf("refdata[%d] = %f\n",i,refdata[i]);
//		}
//
//		refChooseSignal<<<1,4>>>(refdata,max,refline,dbn,dbn_n);
//		__syncthreads();
		break;
	case 1:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 2:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 3:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 4:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 5:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 6:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 7:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 8:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 9:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 10:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 11:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 12:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 13:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 14:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 15:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 16:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 17:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 18:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 19:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 20:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 21:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 22:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 23:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 24:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 25:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 26:
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 27:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 28:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 29:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 30:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	case 31:
		printf("idx = %d\n",idx);
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata);
		__syncthreads();
		memcpy(DeviceDecDataFinsh + finshline * power(2,dbn_n) * idx,refdata,sizeof(double) * finshline * power(2,dbn_n));
		break;
	}
}

__device__ void ref(double* refdata,double* max,int refline,int dbn,int dbn_n,int Begin,int End)
{
	double* refMaxL= new double[2 * dbn]();
	double* refMaxH= new double[2 * dbn]();
	for(int i = 0; i < 2 * dbn; ++i)
	{
		refMaxL[i] = max[2 * 2 * dbn + i];
		refMaxH[i] = max[3 * 2 * dbn + i];
		printf("refMaxL[%d] = %f\n",i,refMaxL[i]);
		printf("refMaxH[%d] = %f\n",i,refMaxH[i]);
	}
	printf("refline = %d\n",refline);
	for(int i = 0; i < 24; ++i)
	{
		printf("refdata[%d] = %f\n",i,refdata[i]);
	}
	for(int DBN_N = 0; DBN_N < 1; ++DBN_N)
	{
		printf("refline = %d\n",refline);
		for(int i = 0; i < 24; ++i)
		{
			printf("refdata[%d] = %f\n",i,refdata[i]);
		}

		int refDsamLen   = 2 * refline + 1;				//重构下采样长度
		int refExLen     = 2 * refline + 4 * dbn - 1;	//重构对称延拓长度
		int refConLen    = 2 * refline + 2 * dbn;		//重构卷积长度
		int refSingalLen = 2 * refline - 2 * dbn + 2;	//重构信号长度

		double* refTransfromDsam = new double[(End - Begin + 1) * refDsamLen]();
		double* refTransfromEx   = new double[(End - Begin + 1) * refExLen]();
		double* refTransfromCon  = new double[(End / 2 - Begin / 2 + 1) * refConLen]();
		double* refTransfromSignal;


		refSignalDSam<<<1,End - Begin + 1>>>(refdata,refTransfromDsam,refTransfromSignal,refDsamLen,Begin,DBN_N,refline);
		__syncthreads();
		printf("refDsamLen = %d\n",refDsamLen);
		for(int i = 0; i < (End - Begin + 1) * refDsamLen; ++i)
		{
			printf("refTransfromDsam[%d] = %f\n",i,refTransfromDsam[i]);
		}
		printf("refExLen = %d\n",refExLen);
		refSignalEx<<<1,End - Begin + 1>>>(refTransfromDsam,refTransfromEx,dbn,refDsamLen,refExLen);
		__syncthreads();
		for(int i = 0; i < (End - Begin + 1) * refExLen; ++i)
		{
			printf("refTransfromEx[%d] = %f\n",i,refTransfromEx[i]);
		}
		delete[] refTransfromDsam;
		//卷积区分高频低频卷积
		//测试卷积效果

		printf("Begin = %d\n",Begin);
		printf("End = %d\n",End);
		if(Begin % 2 == 0)
		{
			if(End % 2 == 0)//第一种情况 Begin和End都是偶数
			{
				printf("执行第一种情况\n");
				refSignalCon<<<1,End / 2 - Begin / 2>>>(refTransfromEx,refTransfromCon,refMaxL,refMaxH,refConLen,refExLen,dbn,Begin,End,0);
				refConEnd<<<1,refConLen>>>(refTransfromEx,refTransfromCon,refMaxL,refConLen,refExLen,dbn,Begin,End);//计算End数据
			}else{//第二种情况Begin是偶数,End是奇数
				printf("执行第二种情况\n");
				refSignalCon<<<1,End / 2 - Begin / 2 + 1>>>(refTransfromEx,refTransfromCon,refMaxL,refMaxH,refConLen,refExLen,dbn,Begin,End,0);
			}
		}else{
			if(End % 2 == 0)//第三种情况Begin是奇数,End是偶数
			{
				printf("执行第三种情况\n");
				refConBegin<<<1,refConLen>>>(refTransfromEx,refTransfromCon,refMaxH,dbn);//计算Begin数据
				refSignalCon<<<1,End / 2 - Begin / 2 - 1>>>(refTransfromEx,refTransfromCon,refMaxL,refMaxH,refConLen,refExLen,dbn,Begin,End,1);
				refConEnd<<<1,refConLen>>>(refTransfromEx,refTransfromCon,refMaxL,refConLen,refExLen,dbn,Begin,End);//计算End数据
			}else{//第四种情况,Begin和End都是奇数
				printf("执行第四种情况\n");
				refConBegin<<<1,refConLen>>>(refTransfromEx,refTransfromCon,refMaxH,dbn);//计算Begin数据
				refSignalCon<<<1,End / 2 - Begin / 2>>>(refTransfromEx,refTransfromCon,refMaxL,refMaxH,refConLen,refExLen,dbn,Begin,End,1);
			}
		}
		delete[] refTransfromEx;
		//打印卷积结果
		printf("refTransfromCon = \n");
		printf("refConLen = %d\n",refConLen);
		for(int i = 0; i < (End / 2 - Begin / 2 + 1) * refConLen; ++i)
		{
			printf("refTransfromCon[%d] = %f\n",i,refTransfromCon[i]);
		}
		//信号选取
		refTransfromSignal = new double[(End / 2 - Begin / 2 + 1) * refSingalLen]();
		refSignal<<<1,End / 2 - Begin / 2 + 1>>>(refTransfromCon,refTransfromSignal,refConLen,refSingalLen,dbn);
		delete[] refTransfromCon;

		printf("refTransfromSignal = \n");
		printf("refSingalLen = %d\n",refSingalLen);
		for(int i = 0; i < (End / 2 - Begin / 2 + 1) * refSingalLen; ++i)
		{
			printf("refTransfromSignal[%d] = %f\n",i,refTransfromSignal[i]);
		}
		Begin = (int)Begin / 2;
		End   = (int)End / 2;
		printf("Begin = %d\n",Begin);
		printf("End = %d\n",End);
	}
	delete[] refMaxL;
	delete[] refMaxH;
}

__global__ void refSignal(double* refTransfromCon,double*refTransfromSignal,int refConLen,int refSingalLen,int dbn)
{
	int refSignalIdx = threadIdx.x;
	refSig<<<1,refSingalLen>>>(refTransfromCon,refTransfromSignal,refConLen,refSingalLen,dbn,refSignalIdx);

}

__global__ void refSig(double* refTransfromCon,double*refTransfromSignal,int refConLen,int refSingalLen,int dbn,int refSignalIdx)
{
	int refSinidx = threadIdx.x;
	refTransfromSignal[refSignalIdx * refSingalLen + refSinidx] = refTransfromCon[refSignalIdx * refConLen + 2 * dbn - 2 + refSinidx];
}

//flag是为了保证Begin和之后的不会重复
__global__ void refConBegin(double* refTransfromEx,double* refTransfromCon,double* refMaxH,int dbn)
{
	int refConidx = threadIdx.x;
	for(int i = 0; i < 2* dbn; ++i)
	{
		refTransfromCon[refConidx] += refTransfromEx[refConidx + i] * refMaxH[i];
	}
}

__global__ void refConEnd(double* refTransfromEx,double* refTransfromCon,double* refMaxL,int refConLen,int refExLen,int dbn,int Begin,int End)
{
	int refConidx = threadIdx.x;
	for(int i = 0; i < 2* dbn; ++i)
	{
		refTransfromCon[(End / 2 - Begin / 2) * refConLen + refConidx] += refTransfromEx[(End - Begin) * refExLen + refConidx + i] * refMaxL[i];
	}
}

__global__ void refCon(double* refTransfromEx,double* refTransfromCon,double* refMaxL,double* refMaxH,int refConLen,int refExLen,int dbn,int refConBlockIdx,int flag)
{
	int refConidx = threadIdx.x;

	for(int i = 0; i < 2* dbn; ++i)
	{
		refTransfromCon[(refConBlockIdx + flag) * refConLen + refConidx] += refTransfromEx[2 * refConBlockIdx * refExLen + flag  * refExLen + refConidx + i] * refMaxL[i]
		                                                        + refTransfromEx[(2 * refConBlockIdx + 1) * refExLen  + flag * refExLen + refConidx + i] * refMaxH[i];
	}
}

__global__ void refSignalCon(double* refTransfromEx,double* refTransfromCon,double* refMaxL,double* refMaxH,int refConLen,int refExLen,int dbn,int Begin,int End,int flag)
{
	int refConBlockIdx = threadIdx.x;

	refCon<<<1,refConLen>>>(refTransfromEx,refTransfromCon,refMaxL,refMaxH,refConLen,refExLen,dbn,refConBlockIdx,flag);
}

__device__ void refEx(double* refTransfromDsam,double* refTransfromEx,int dbn,int refDsamLen,int refExLen,int refExidx)
{
	for(int i = 0; i < 2 * dbn - 1; ++i)
	{
		refTransfromEx[refExidx * refExLen + i] 				       = refTransfromDsam[refExidx * refDsamLen + 2 * dbn -2 - i];
		refTransfromEx[refExidx * refExLen + refDsamLen + 2 * dbn -1 + i] = refTransfromDsam[refExidx * refDsamLen + refDsamLen -1 - i];
	}
	for(int i = 0; i < refDsamLen; ++i)
	{
		refTransfromEx[refExidx * refExLen + 2 * dbn - 1 + i] = refTransfromDsam[refExidx * refDsamLen + i];
	}
}

__global__ void refSignalEx(double* refTransfromDsam,double* refTransfromEx,int dbn,int refDsamLen,int refExLen)
{
	int refExidx = threadIdx.x;
	refEx(refTransfromDsam,refTransfromEx,dbn,refDsamLen,refExLen,refExidx);
}

__global__ void refDSam(double* refdata,double* refTransfromDsam,double* refTransfromSignal,int refDsamLen,int Begin,int DBN_N,int refline,int refDsamidx)
{
	int refDidx = threadIdx.x;
	if(DBN_N == 0)
	{
		refTransfromDsam[refDsamidx * refDsamLen + 2 * refDidx + 1] = refdata[(refDsamidx + Begin) * refline + refDidx];
	}else{
		refTransfromDsam[refDsamidx * refDsamLen + 2 * refDidx + 1] = refTransfromSignal[refDsamidx * refDsamLen + refDidx];
	}
}

__global__ void refSignalDSam(double* refdata,double* refTransfromDsam,double* refTransfromSignal,int refDsamLen,int Begin,int DBN_N,int refline)
{
	int refDsamidx = threadIdx.x;
	refDSam<<<1,refline>>>(refdata,refTransfromDsam,refTransfromSignal,refDsamLen,Begin,DBN_N,refline,refDsamidx);
	__syncthreads();
}

/*
 * 选择波形一共四种波形,Alpha,Beta,Delta,Theta,指定开四个线程,每个线程代表一个波形的重构,
 * 0线程代表Alpha,1线程代表Beta,2线程代表Delta,3线程代表Theta
 *
 * */
__global__ void refChooseSignal(double* refdata,double* max,int refline,int dbn,int dbn_n)
{
	int refchSigidx = threadIdx.x;

	switch(refchSigidx)
	{
	case 0:
		/*
		 * Alpha波的起始频率为7.81Hz终止信号为13.28Hz
		 *
		 * */
		printf("选择Alpha波.\n");
//		int AlphaBegin = static_cast<int>(floor(7.81 * power(2,dbn_n) / 64));
//		int AlphaEnd   = static_cast<int>(floor(13.28 * power(2,dbn_n) / 64));
		int AlphaBegin = 0;
		int AlphaEnd = 1;
		printf("AlphaBegin = %d\n",AlphaBegin);
		printf("AlphaEnd = %d\n",AlphaEnd);
		ref(refdata,max,refline,dbn,dbn_n,AlphaBegin,AlphaEnd);


		break;
	case 1:
		/*
		 * Beta波的起始频率为13.28Hz终止信号为30.47Hz
		 *
		 * */
		printf("选择Beta波.\n");
		int BetaBegin = static_cast<int>(floor(13.28 * power(2,dbn_n) / 64));
		int BetaEnd   = static_cast<int>(floor(30.47 * power(2,dbn_n) / 64));
		printf("BetaBegin = %d\n",BetaBegin);
		printf("BetaEnd = %d\n",BetaEnd);
		break;
	case 2:
		/*
		 * Delta波的起始频率为0.78Hz终止信号为3.91Hz
		 *
		 * */
		printf("选择Delta波.\n");
		int DeltaBegin = static_cast<int>(floor(0.78 * power(2,dbn_n) / 64));
		int DeltaEnd   = static_cast<int>(floor(3.91 * power(2,dbn_n) / 64));
		printf("DeltaBegin = %d\n",DeltaBegin);
		printf("DeltaEnd = %d\n",DeltaEnd);
		break;
	case 3:
		/*
		 *Theta波的起始频率为3.91Hz终止信号为7.81Hz
		 *
		 * */
		printf("选择Theta波.\n");
		int ThetaBegin = static_cast<int>(floor(3.91 * power(2,dbn_n) / 64));
		int ThetaEnd   = static_cast<int>(floor(7.81 * power(2,dbn_n) / 64));
		printf("ThetaBegin = %d\n",ThetaBegin);
		printf("ThetaEnd = %d\n",ThetaEnd);
		break;
	}


}

__global__ void GPU1(double* d_a,int row,int line,int dbn,int dbn_n)
{


}

__device__ void printMax(double* max,int dbn)
{
	printf("GPU打印\n");
    for(int i = 0; i < 4; ++i)
    {
        for(int j = 0; j < 2 * dbn; ++j)
        {
        	printf("%f\t",max[i * 2 * dbn + j]);
        }
        printf("\n");
    }
}

__device__ void printData(double* buffer,int row,int line)
{
    for(int i = 0; i < row; ++i)
    {
        for(int j = 0; j < line; ++j)
        {
        	printf("%d\t",buffer[i * line + j]);
        }
        printf("\n");
    }
}
// 这个函数用于开线程延拓线程开一组的就可以其他组的用line来区分
__global__ void decEx(double* dVectorUpSam,int line,int dbn,double* dVectorEx)
{
	int decExidx = threadIdx.x;//这个线程号最多只有一组的线程号
	for(int i = 0; i < 2 * dbn - 1; ++i)
	{
		dVectorEx[decExidx  * (line + 4 * dbn - 2) + i] 					 = dVectorUpSam[decExidx * line + 2 * dbn -2 - i];
		dVectorEx[decExidx  * (line + 4 * dbn - 2) + 2 * dbn + line - 1 + i] = dVectorUpSam[decExidx * line + line -1 - i];
	}
	for(int i = 0; i < line; ++i)
	{
		dVectorEx[decExidx  * (line + 4 * dbn - 2) + 2 * dbn - 1 + i] = dVectorUpSam[decExidx  * line + i];
	}
}

__device__ void decTransfromEx(double* d_data,double* dVectorUpSam,int line,int idx,int dbn,int decExLen,double* dVectorEx,int DBN_N)
{
	if(DBN_N == 0)
	{
		for(int i = 0; i < 2 * dbn - 1; ++i)
		{
			dVectorEx[i] 					  = d_data[idx * line + 2 * dbn -2 - i];
			dVectorEx[2 * dbn + line - 1 + i] = d_data[idx * line + line -1 - i];
		}
		for(int i = 0; i < line; ++i)
		{
			dVectorEx[2 * dbn - 1 + i] = d_data[idx * line + i];
		}
	}else{
		decEx<<<1,power(2,DBN_N)>>>(dVectorUpSam,line,dbn,dVectorEx);
	}
	return ;
}

__global__ void dataCON(double* dVectorEx,double* sdecMaxL,double* sdecMaxH,int dbn,int decCONLen,double* dVectorCON,int decCONidx,int decExLen,int DBN_N)
{
	int iidx = blockIdx.x * blockDim.x + threadIdx.x;
	if(iidx >= decCONLen)
	{
	}else{
		for(int i = 0; i < 2* dbn; ++i)
		{
			dVectorCON[2 * decCONidx * decCONLen + iidx] 	   += dVectorEx[decCONidx * decExLen + iidx + i] * sdecMaxL[i];
			dVectorCON[(2 * decCONidx + 1) * decCONLen + iidx] += dVectorEx[decCONidx * decExLen + iidx + i] * sdecMaxH[i];
		}
	}
}

__global__ void decTransfromCON(double* dVectorEx,double* sdecMaxL,double*sdecMaxH,int decCONLen,int dbn,double* dVectorCON,int decExLen,int DBN_N)
{
	int decCONidx = threadIdx.x;
	int girdconIdx  = 0;
	int blockconIdx = 0;
	if(0 == decCONLen / 1000)
	{
		girdconIdx = 1;
		blockconIdx = decCONLen;
	}else{
		girdconIdx  = decCONLen / 1000 + 1;
		blockconIdx = 1000;
	}
	dataCON<<<girdconIdx,blockconIdx>>>(dVectorEx,sdecMaxL,sdecMaxH,dbn,decCONLen,dVectorCON,decCONidx,decExLen,DBN_N);
	__syncthreads();
}

__global__ void decUpSam(double* dVectorCON,double*dVectorUpSam,int decCONLen,int decUpSamLen,int DBN_N,int decUpidx)
{
	int iUpdx = blockIdx.x * blockDim.x + threadIdx.x;
	dVectorUpSam[decUpSamLen * decUpidx + iUpdx] = dVectorCON[decCONLen * decUpidx + 2 * iUpdx + 1];
}

__global__ void decTransfromUpSam(double* dVectorCON,double*dVectorUpSam,int decCONLen,int decUpSamLen,int DBN_N)
{
	int decUpidx = threadIdx.x;
	int gridUpsamIdx  = 0;
	int blockUpsamIdx = 0;
	if(decUpSamLen >= 1000)
	{
		gridUpsamIdx  = decUpSamLen / 1000 + 1;
		blockUpsamIdx = 1000;
	}else{
		gridUpsamIdx  = 1;
		blockUpsamIdx = decUpSamLen;
	}
	decUpSam<<<gridUpsamIdx,blockUpsamIdx>>>(dVectorCON,dVectorUpSam,decCONLen,decUpSamLen,DBN_N,decUpidx);
	__syncthreads();
}

__device__ void dec(double* d_a,double* max,int row,int line,int dbn,int idx,int dbn_n,double*& refdata)
{
	/*分解过程*/
	double* sdecMaxL = new double[2 * dbn]();
	double* sdecMaxH = new double[2 * dbn]();
	for(int i = 0; i < 2 * dbn; ++i)
	{
		sdecMaxL[i] = max[0 * 2 * dbn + i];
		sdecMaxH[i] = max[1 * 2 * dbn + i];
	}
	for(int DBN_N = 0; DBN_N < dbn_n; ++DBN_N)
	{
		int decExLen    = line + 4 * dbn - 2;//对称延拓长度
		int decCONLen   = line + 2 * dbn - 1;//分解卷积长度
		int decUpSamLen = (line + 2* dbn -1) / 2;//上采样长度
		double* dVectorEx  = new double[power(2,DBN_N) * decExLen]();
		double* dVectorCON = new double[power(2,DBN_N + 1) * decCONLen]();		//  卷积和内存
		double* dVectorUpSam; 													//  上采样内存
		decTransfromEx(d_a,dVectorUpSam,line,idx,dbn,decExLen,dVectorEx,DBN_N);//延拓
		decTransfromCON<<<1,power(2,DBN_N)>>>(dVectorEx,sdecMaxL,sdecMaxH,decCONLen,dbn,dVectorCON,decExLen,DBN_N);//卷积
		__syncthreads();
		//只能在这里申请内存,在开头申请内存第二次循环数据为空了.
		if(DBN_N != 0)
		{
			delete[]dVectorUpSam;
		}
		dVectorUpSam = new double[power(2,DBN_N + 1) * decUpSamLen]();
		decTransfromUpSam<<<1,power(2,DBN_N + 1)>>>(dVectorCON,dVectorUpSam,decCONLen,decUpSamLen,DBN_N);
		__syncthreads();
		//更新系数,进行下一层分解
		line = decUpSamLen;
		delete[]dVectorEx;
		delete[]dVectorCON;
		if(dbn_n - 1 == DBN_N)//重构信号长度
		{
			refdata = dVectorUpSam;
			memcpy(refdata + line * power(2,dbn_n) * idx,dVectorUpSam,sizeof(double) * line * power(2,dbn_n));
			for(int i = 0; i < line; ++i)
			{
				printf("dVectorUpSam[%d] =  %f\n",i,dVectorUpSam[i]);
				printf("refdata[%d] =  %f\n",i,refdata[i]);
			}
			delete[]dVectorUpSam;
		}
	}
	delete[]sdecMaxL;
	delete[]sdecMaxH;
}
