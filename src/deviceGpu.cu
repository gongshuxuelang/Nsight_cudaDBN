#include "hip/hip_runtime.h"
#include "head.h"
/*
 * 说明,GPU分出一个文件的向量,例如32个行向量,需要分出32个线程.
 * 进入GPU中每一个线程又需要干不同的任务,一个行向量要进行DBN_N次分解和重构
 * 每一次分解要比上一次的行向量多2倍,总体上就是2的DBN_N次幂的关系.
 * 我需要的做的就是理清楚一个行向量做出2维数据,用并行算法写出代码.避免串行代码的问题,既然使用了GPU那就不要浪费GPU的计算性能.
 * */
__device__ int power(int base, int exponent)
{
	int result = 1;
	if(exponent == 0)
	{
		return result;
	}
	for (int i = 0; i < exponent; ++i)
	{
		result *= base;
	}

	return result;
}
//接收的数据为32*7681,32行互不相干。开32个线程来处理这个矩阵。d_data是数据row是行，line是列，DBN_N是分解层数
__global__ void GPU0(double* d_a,double* max,int row,int line,int dbn,int dbn_n)
{
	int idx = threadIdx.x;
	switch(idx)
	{
	case 0:
		int refline = 0;
		double* refdata;
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata,refline);
		__syncthreads();
		/*重构部分*/
		printf("refline = %d\n",refline);
		printf("power(2,dbn_n) * refline = %d\n",power(2,dbn_n) * refline);
		for(int i = 0; i < power(2,dbn_n) * refline;++i)
		{
			printf("refdata[%d] = %f\n",i,refdata[i]);
		}
		refChooseSignal<<<1,4>>>(refdata,max,refline,dbn,dbn_n);
		__syncthreads();
		break;
	case 1:

		__syncthreads();
		break;
	case 2:

		__syncthreads();
		break;
	case 3:

		__syncthreads();
		break;
	}
}

__device__ void ref(double* refdata,double* max,int refline,int dbn,int dbn_n,int Begin,int End)
{
	double* refMaxL= new double[2 * dbn]();
	double* refMaxH= new double[2 * dbn]();
	for(int i = 0; i < 2 * dbn; ++i)
	{
		refMaxL[i] = max[2 * 2 * dbn + i];
		refMaxH[i] = max[3 * 2 * dbn + i];
		printf("refMaxL[%d] = %f\n",i,refMaxL[i]);
		printf("refMaxH[%d] = %f\n",i,refMaxH[i]);
	}

	for(int DBN_N = 0; DBN_N < 1; ++DBN_N)
	{
		int refDsamLen   = 2 * refline + 1;				//重构下采样长度
		int refExLen     = 2 * refline + 4 * dbn - 1;	//重构对称延拓长度
		int refConLen    = 2 * refline + 2 * dbn;		//重构卷积长度
		int refSingalLen = 2 * refline - 2 * dbn + 2;	//重构信号长度

		double* refTransfromDsam = new double[(End - Begin + 1) * refDsamLen]();
		double* refTransfromEx   = new double[(End - Begin + 1) * refExLen]();
		double* refTransfromCon  = new double[(End / 2 + Begin / 2 + 1) * refConLen]();
		double* refTransfromSignal;


		refSignalDSam<<<1,End - Begin + 1>>>(refdata,refTransfromDsam,refTransfromSignal,refDsamLen,Begin,DBN_N,refline);
		__syncthreads();
		printf("refDsamLen = %d\n",refDsamLen);
		for(int i = 0; i < (End - Begin + 1) * refDsamLen; ++i)
		{
			printf("refTransfromDsam[%d] = %f\n",i,refTransfromDsam[i]);
		}
		printf("refExLen = %d\n",refExLen);
		refSignalEx<<<1,End - Begin + 1>>>(refTransfromDsam,refTransfromEx,dbn,refDsamLen,refExLen);
		__syncthreads();
		for(int i = 0; i < (End - Begin + 1) * refExLen; ++i)
		{
			printf("refTransfromEx[%d] = %f\n",i,refTransfromEx[i]);
		}

	}
	delete[] refMaxL;
	delete[] refMaxH;
}

__global__ void refCon(double* refTransfromEx,double* refTransfromCon,double* refMaxL,double* refMaxH,int refConLen,int refExLen,int dbn,int Begin,int End)
{
	int refConidx = threadIdx.x;

	for(int i = 0; i < 2* dbn; ++i)
	{


	}
}
__device__ void refSignalCon(double* refTransfromEx,double* refTransfromCon,double* refMaxL,double* refMaxH,int refConLen,int refExLen,int dbn,int Begin,int End)
{
	if(Begin % 2 == 0)
	{
		if(End % 2 == 0)//第一种情况 Begin和End都是偶数
		{

		}else{//第二种情况Begin是偶数,End是奇数


		}

	}else{
		if(End % 2 == 0)//第三种情况Begin是奇数,End是偶数
		{

		}else{//第四种情况,Begin和End都是奇数

		}
	}
}


__device__ void refEx(double* refTransfromDsam,double* refTransfromEx,int dbn,int refDsamLen,int refExLen,int refExidx)
{
	for(int i = 0; i < 2 * dbn - 1; ++i)
	{
		refTransfromEx[refExidx * refExLen + i] 				       = refTransfromDsam[refExidx * refDsamLen + 2 * dbn -2 - i];
		refTransfromEx[refExidx * refExLen + refDsamLen + 2 * dbn -1 + i] = refTransfromDsam[refExidx * refDsamLen + refDsamLen -1 - i];
	}
	for(int i = 0; i < refDsamLen; ++i)
	{
		refTransfromEx[refExidx * refExLen + 2 * dbn - 1 + i] = refTransfromDsam[refExidx * refDsamLen + i];
	}
}

__global__ void refSignalEx(double* refTransfromDsam,double* refTransfromEx,int dbn,int refDsamLen,int refExLen)
{
	int refExidx = threadIdx.x;
	refEx(refTransfromDsam,refTransfromEx,dbn,refDsamLen,refExLen,refExidx);
}

__global__ void refDSam(double* refdata,double* refTransfromDsam,double* refTransfromSignal,int refDsamLen,int Begin,int DBN_N,int refline,int refDsamidx)
{
	int refDidx = threadIdx.x;
	if(DBN_N == 0)
	{
		refTransfromDsam[refDsamidx * refDsamLen + 2 * refDidx + 1] = refdata[(refDsamidx + Begin) * refline + refDidx];
	}else{
		refTransfromDsam[refDsamidx * refDsamLen + 2 * refDidx + 1] = refTransfromSignal[refDsamidx * refDsamLen + refDidx];
	}
}

__global__ void refSignalDSam(double* refdata,double* refTransfromDsam,double* refTransfromSignal,int refDsamLen,int Begin,int DBN_N,int refline)
{
	int refDsamidx = threadIdx.x;
	refDSam<<<1,refline>>>(refdata,refTransfromDsam,refTransfromSignal,refDsamLen,Begin,DBN_N,refline,refDsamidx);
	__syncthreads();
}

/*
 * 选择波形一共四种波形,Alpha,Beta,Delta,Theta,指定开四个线程,每个线程代表一个波形的重构,
 * 0线程代表Alpha,1线程代表Beta,2线程代表Delta,3线程代表Theta
 *
 * */
__global__ void refChooseSignal(double* refdata,double* max,int refline,int dbn,int dbn_n)
{
	int refchSigidx = threadIdx.x;

	switch(refchSigidx)
	{
	case 0:
		/*
		 * Alpha波的起始频率为7.81Hz终止信号为13.28Hz
		 *
		 * */
		printf("选择Alpha波.\n");
//		int AlphaBegin = static_cast<int>(floor(7.81 * power(2,dbn_n) / 64));
//		int AlphaEnd   = static_cast<int>(floor(13.28 * power(2,dbn_n) / 64));
		int AlphaBegin = 1;
		int AlphaEnd = 5;
		printf("AlphaBegin = %d\n",AlphaBegin);
		printf("AlphaEnd = %d\n",AlphaEnd);
		ref(refdata,max,refline,dbn,dbn_n,AlphaBegin,AlphaEnd);


		break;
	case 1:
		/*
		 * Beta波的起始频率为13.28Hz终止信号为30.47Hz
		 *
		 * */
		printf("选择Beta波.\n");
		int BetaBegin = static_cast<int>(floor(13.28 * power(2,dbn_n) / 64));
		int BetaEnd   = static_cast<int>(floor(30.47 * power(2,dbn_n) / 64));
		printf("BetaBegin = %d\n",BetaBegin);
		printf("BetaEnd = %d\n",BetaEnd);
		break;
	case 2:
		/*
		 * Delta波的起始频率为0.78Hz终止信号为3.91Hz
		 *
		 * */
		printf("选择Delta波.\n");
		int DeltaBegin = static_cast<int>(floor(0.78 * power(2,dbn_n) / 64));
		int DeltaEnd   = static_cast<int>(floor(3.91 * power(2,dbn_n) / 64));
		printf("DeltaBegin = %d\n",DeltaBegin);
		printf("DeltaEnd = %d\n",DeltaEnd);
		break;
	case 3:
		/*
		 *Theta波的起始频率为3.91Hz终止信号为7.81Hz
		 *
		 * */
		printf("选择Theta波.\n");
		int ThetaBegin = static_cast<int>(floor(3.91 * power(2,dbn_n) / 64));
		int ThetaEnd   = static_cast<int>(floor(7.81 * power(2,dbn_n) / 64));
		printf("ThetaBegin = %d\n",ThetaBegin);
		printf("ThetaEnd = %d\n",ThetaEnd);
		break;
	}


}

__global__ void GPU1(double* d_a,int row,int line,int dbn,int dbn_n)
{


}

__device__ void printMax(double* max,int dbn)
{
	printf("GPU打印\n");
    for(int i = 0; i < 4; ++i)
    {
        for(int j = 0; j < 2 * dbn; ++j)
        {
        	printf("%f\t",max[i * 2 * dbn + j]);
        }
        printf("\n");
    }
}

__device__ void printData(double* buffer,int row,int line)
{
    for(int i = 0; i < row; ++i)
    {
        for(int j = 0; j < line; ++j)
        {
        	printf("%d\t",buffer[i * line + j]);
        }
        printf("\n");
    }
}
// 这个函数用于开线程延拓线程开一组的就可以其他组的用line来区分
__global__ void decEx(double* dVectorUpSam,int line,int dbn,double* dVectorEx)
{
	int decExidx = threadIdx.x;//这个线程号最多只有一组的线程号

	for(int i = 0; i < 2 * dbn - 1; ++i)
	{
		dVectorEx[decExidx  * (line + 4 * dbn - 2) + i] 					 = dVectorUpSam[decExidx * line + 2 * dbn -2 - i];
		dVectorEx[decExidx  * (line + 4 * dbn - 2) + 2 * dbn + line - 1 + i] = dVectorUpSam[decExidx * line + line -1 - i];
	}
	for(int i = 0; i < line; ++i)
	{
		dVectorEx[decExidx  * (line + 4 * dbn - 2) + 2 * dbn - 1 + i] = dVectorUpSam[decExidx  * line + i];
	}
}

__device__ void decTransfromEx(double* d_data,double* dVectorUpSam,int line,int idx,int dbn,int decExLen,double* dVectorEx,int DBN_N)
{
	if(DBN_N == 0)
	{
		for(int i = 0; i < 2 * dbn - 1; ++i)
		{
			dVectorEx[i] 					  = d_data[idx * line + 2 * dbn -2 - i];
			dVectorEx[2 * dbn + line - 1 + i] = d_data[idx * line + line -1 - i];
		}
		for(int i = 0; i < line; ++i)
		{
			dVectorEx[2 * dbn - 1 + i] = d_data[idx * line + i];
		}
	}else{
		decEx<<<1,power(2,DBN_N)>>>(dVectorUpSam,line,dbn,dVectorEx);
	}
	return ;
}

__global__ void dataCON(double* dVectorEx,double* sdecMaxL,double* sdecMaxH,int dbn,int decCONLen,double* dVectorCON,int decCONidx,int decExLen,int DBN_N)
{
	int iidx = threadIdx.x;
	for(int i = 0; i < 2* dbn; ++i)
	{
		dVectorCON[2 * decCONidx * decCONLen + iidx] += dVectorEx[decCONidx * decExLen + iidx + i] * sdecMaxL[i];
		dVectorCON[(2 * decCONidx + 1) * decCONLen + iidx] += dVectorEx[decCONidx * decExLen + iidx + i] * sdecMaxH[i];
	}
}

__global__ void decTransfromCON(double* dVectorEx,double* sdecMaxL,double*sdecMaxH,int decCONLen,int dbn,double* dVectorCON,int decExLen,int DBN_N)
{
	int decCONidx = threadIdx.x;
	dataCON<<<1,decCONLen>>>(dVectorEx,sdecMaxL,sdecMaxH,dbn,decCONLen,dVectorCON,decCONidx,decExLen,DBN_N);
	__syncthreads();
}

__global__ void decUpSam(double* dVectorCON,double*dVectorUpSam,int decCONLen,int decUpSamLen,int DBN_N,int decUpidx)
{
	int iUpdx = threadIdx.x;
	dVectorUpSam[decUpSamLen * decUpidx + iUpdx] = dVectorCON[decCONLen * decUpidx + 2 * iUpdx + 1];
}

__global__ void decTransfromUpSam(double* dVectorCON,double*dVectorUpSam,int decCONLen,int decUpSamLen,int DBN_N)
{
	int decUpidx = threadIdx.x;
	decUpSam<<<1,decUpSamLen>>>(dVectorCON,dVectorUpSam,decCONLen,decUpSamLen,DBN_N,decUpidx);
	__syncthreads();
}

__device__ void dec(double* d_a,double* max,int row,int line,int dbn,int idx,int dbn_n,double*& refdata,int& refline)
{
	/*分解过程*/
	double* sdecMaxL = new double[2 * dbn]();
	double* sdecMaxH = new double[2 * dbn]();
	for(int i = 0; i < 2 * dbn; ++i)
	{
		sdecMaxL[i] = max[0 * 2 * dbn + i];
		sdecMaxH[i] = max[1 * 2 * dbn + i];
		printf("sdecMaxL[%d] = %f\n",i,sdecMaxL[i]);
		printf("sdecMaxH[%d] = %f\n",i,sdecMaxH[i]);
	}
	for(int DBN_N = 0; DBN_N < dbn_n; ++DBN_N)
	{
		int decExLen    = line + 4 * dbn - 2;//对称延拓长度
		int decCONLen   = line + 2 * dbn - 1;//分解卷积长度
		int decUpSamLen = (line + 2* dbn -1) / 2;//上采样长度

		double* dVectorEx 	  = new double[power(2,DBN_N) * decExLen]();
		double* dVectorCON    = new double[power(2,DBN_N + 1) * decCONLen]();		//  卷积和内存
		double* dVectorUpSam; 													//  上采样内存



		decTransfromEx(d_a,dVectorUpSam,line,idx,dbn,decExLen,dVectorEx,DBN_N);//延拓

		printf("decExLen = %d\n",decExLen);
		for(int i = 0; i < power(2,DBN_N) * decExLen;++i)
		{
			printf("dVectorEx[%d] = %f\n",i,dVectorEx[i]);
		}
		printf("\n");


		decTransfromCON<<<1,power(2,DBN_N)>>>(dVectorEx,sdecMaxL,sdecMaxH,decCONLen,dbn,dVectorCON,decExLen,DBN_N);//卷积
		__syncthreads();
		delete[]dVectorEx;

		printf("decCONLen = %d\n",decCONLen);
		for(int i = 0; i < power(2,DBN_N + 1) * decCONLen;++i)
		{
			printf("dVectorCON[%d] = %f\n",i,dVectorCON[i]);
		}
		//只能在这里申请内存,在开头申请内存第二次循环数据为空了.
		dVectorUpSam = new double[power(2,DBN_N + 1) * decUpSamLen]();
		decTransfromUpSam<<<1,power(2,DBN_N + 1)>>>(dVectorCON,dVectorUpSam,decCONLen,decUpSamLen,DBN_N);
		__syncthreads();
		delete[]dVectorCON;

		printf("decUpSamLen = %d\n",decUpSamLen);
		for(int i = 0; i < power(2,DBN_N + 1) * decUpSamLen;++i)
		{
			printf("dVectorUpSam[%d] = %f\n",i,dVectorUpSam[i]);
		}
		//更新系数,进行下一层分解
		line = decUpSamLen;
		if(DBN_N == dbn_n - 1)//重构信号长度
		{
			refline = line;
			refdata = new double[power(2,dbn_n)*refline];
			refdata = dVectorUpSam;
			delete[]dVectorUpSam;
		}
	}
	delete[]sdecMaxL;
	delete[]sdecMaxH;
}


