#include "hip/hip_runtime.h"
#include "head.h"


//接收的数据为32*7681,32行互不相干。开32个线程来处理这个矩阵。d_data是数据row是行，line是列，DBN_N是分解层数
__global__ void GPU0(double* d_a,double* max,int row,int line,int dbn,int dbn_n)
{
	int idx = threadIdx.x;
	switch(idx)
	{
	case 0:
		dec(d_a,max,row,line,dbn,idx);

		break;
	case 1:
		break;
	case 2:
		break;
	case 3:
		break;
	}
}

__global__ void GPU1(double* d_a,int row,int line,int dbn,int dbn_n)
{


}

__device__ void printMax(double* max,int dbn)
{
	printf("GPU打印\n");
    for(int i = 0; i < 4; ++i)
    {
        for(int j = 0; j < 2 * dbn; ++j)
        {
        	printf("%f\t",max[i * 2 * dbn + j]);
        }
        printf("\n");
    }
}

__device__ void printData(double* buffer,int row,int line)
{
    for(int i = 0; i < row; ++i)
    {
        for(int j = 0; j < line; ++j)
        {
        	printf("%d\t",buffer[i * line + j]);
        }
        printf("\n");
    }
}
__device__ void decTransfromEx(double* d_data,int line,int idx,int dbn,int decExLen,double* dVectorEx)
{
	for(int i = 0; i < line; ++i)
	{
		printf("d_data  = %f ",d_data[i]);
	}
	printf("\n");
	for(int i = 0; i < 2 * dbn - 1; ++i)
	{
		dVectorEx[i] = d_data[idx * line + 2 * dbn -2 - i];
		dVectorEx[2 * dbn + line - 1 + i] = d_data[idx * line + line -1 - i];
	}
	for(int i = 0; i < line; ++i)
	{
		dVectorEx[2 * dbn - 1 + i] = d_data[idx * line + i];
	}
	for(int i = 0; i < decExLen; ++i)
	{
		printf("dVectorEx = %f ",dVectorEx[i]);
	}
	printf("\n");
	return ;
}
__global__ void dataCON(double* dVectorEx,double* sdecMaxL,double* sdecMaxH,int dbn,int decCONLen,double* dVectorCONL,double* dVectorCONH,int idx,int decExLen)
{
	double tempL = 0;
	double tempH = 0;
	int iidx = threadIdx.x;
	for(int i = 0; i < 2* dbn; ++i)
	{
		tempL += dVectorEx[iidx + i] * sdecMaxL[i];
		tempH += dVectorEx[iidx + i] * sdecMaxH[i];
	}
	dVectorCONL[iidx] = tempL;
	dVectorCONH[iidx] = tempH;
}
__device__ void decTransfromCON(double* dVectorEx,double* max,int line,int idx,int decCONLen,int dbn,double* dVectorCONL,double* dVectorCONH,int decExLen)
{
	double* sdecMaxL= new double[2 * dbn];
	double* sdecMaxH= new double[2 * dbn];

	for(int i = 0; i < 2 * dbn; ++i)
	{
		sdecMaxL[i] = max[0 * 2 * dbn + i];
		sdecMaxH[i] = max[1 * 2 * dbn + i];
		printf("sdecMaxL = %f\t",sdecMaxL[i]);
		printf("\n");
		printf("sdecMaxH = %f\t",sdecMaxH[i]);
		printf("\n");
	}
	dataCON<<<1,decCONLen>>>(dVectorEx,sdecMaxL,sdecMaxH,dbn,decCONLen,dVectorCONL,dVectorCONH,idx,decExLen);
}
__global__ void decUpSam(double* dVectorCONL,double* dVectorCONH,double*dVectorUpSamH,double*dVectorUpSamL)
{
	int iUdx = threadIdx.x;
	dVectorUpSamH[iUdx] = dVectorCONH[2 * iUdx + 1];
	dVectorUpSamL[iUdx] = dVectorCONL[2 * iUdx + 1];
}
__device__ void dec(double* d_a,double* max,int row,int line,int dbn,int idx)
{
	int decExLen    = line + 4 * dbn - 2;//对称延拓长度
	int decCONLen   = line + 2 * dbn - 1;//分解卷积长度
	int decUpSamLen = (line + 2* dbn -1) / 2;//上采样长度

	double* dVectorEx 	  = new double[decExLen];
	double* dVectorCONL   = new double[decCONLen];
	double* dVectorCONH   = new double[decCONLen];
	double* dVectorUpSamH = new double[decUpSamLen];
	double* dVectorUpSamL = new double[decUpSamLen];
	double* dVectorUpSam  = new double[2 * decUpSamLen];
	decTransfromEx(d_a,line,idx,dbn,decExLen,dVectorEx);//延拓
	for(int i = 0; i < decExLen;++i)
	{
		printf("case:dVectorEx = %f\t",dVectorEx[i]);
	}
	printf("\n");
	decTransfromCON(dVectorEx,max,line,idx,decCONLen,dbn,dVectorCONL,dVectorCONH,decExLen);//卷积

	hipFree(dVectorEx);
	printf("CON=\n");
	printf("decCONLen = %d\n",decCONLen);
	for(int i = 0; i < decCONLen;++i)
	{
		printf("dVectorCONL[%d] = %f\n  ",i,dVectorCONL[i]);
		printf("dVectorCONH[%d] = %f\n  ",i,dVectorCONH[i]);
	}
	decUpSam<<<1,decUpSamLen>>>(dVectorCONL,dVectorCONH,dVectorUpSamH,dVectorUpSamL);

	hipFree(dVectorCONL);
	hipFree(dVectorCONH);
	printf("UpSam=\n");
	printf("decUpSamLen = %d\n",decUpSamLen);
	for(int i = 0; i < decUpSamLen;++i)
	{
		printf("dVectorUpSamL[%d] = %f\n  ",i,dVectorUpSamL[i]);
		printf("dVectorUpSamH[%d] = %f\n  ",i,dVectorUpSamH[i]);
		dVectorUpSam[i] 			  = dVectorUpSamL[i];
		dVectorUpSam[decUpSamLen + i] = dVectorUpSamH[i];
	}
	hipFree(dVectorUpSamL);
	hipFree(dVectorUpSamH);
	for(int i = 0; i < 2 * decUpSamLen;++i)
	{
		printf("dVectorUpSam[%d] = %f\n  ",i,dVectorUpSam[i]);
	}

}
