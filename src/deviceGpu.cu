#include "hip/hip_runtime.h"
#include "head.h"
/*
 * 说明,GPU分出一个文件的向量,例如32个行向量,需要分出32个线程.
 * 进入GPU中每一个线程又需要干不同的任务,一个行向量要进行DBN_N次分解和重构
 * 每一次分解要比上一次的行向量多2倍,总体上就是2的DBN_N次幂的关系.
 * 我需要的做的就是理清楚一个行向量做出2维数据,用并行算法写出代码.避免串行代码的问题,既然使用了GPU那就不要浪费GPU的计算性能.
 * */
__device__ int power(int base, int exponent)
{
	int result = 1;
	if(exponent == 0)
	{
		return result;
	}
	for (int i = 0; i < exponent; ++i)
	{
		result *= base;
	}

	return result;
}
//接收的数据为32*7681,32行互不相干。开32个线程来处理这个矩阵。d_data是数据row是行，line是列，DBN_N是分解层数
__global__ void GPU0(double* d_a,double* max,int row,int line,int dbn,int dbn_n)
{
	int idx = threadIdx.x;
	switch(idx)
	{
	case 0:
		int refline = 0;
		double* refdata;
		dec(d_a,max,row,line,dbn,idx,dbn_n,refdata,refline);
		__syncthreads();
		/*重构部分*/
		printf("refline = %d\n",refline);
		printf("power(2,dbn_n) * refline = %d\n",power(2,dbn_n) * refline);
		for(int i = 0; i < power(2,dbn_n) * refline;++i)
		{
			printf("refdata[%d] = %f\n",i,refdata[i]);
		}
		ref(refdata,max,refline,dbn,dbn_n);
		break;
	case 1:

		__syncthreads();
		break;
	case 2:

		__syncthreads();
		break;
	case 3:

		__syncthreads();
		break;
	}
}

__device__ void ref(double* refdata,double* max,int refline,int dbn,int dbn_n)
{
	refChooseSignal<<<1,4>>>(refdata,max,refline,dbn,dbn_n);
}

/*
 * 选择波形一共四种波形,Alpha,Beta,Delta,Theta,指定开四个线程,每个线程代表一个波形的重构,
 * 0线程代表Alpha,1线程代表Beta,2线程代表Delta,3线程代表Theta
 *
 * */
__global__ void refChooseSignal(double* refdata,double* max,int refline,int dbn,int dbn_n)
{
	int refchSigidx = threadIdx.x;

	switch(refchSigidx)
	{
	case 0:
		printf("refChooseSignal 0 = %d\n",refchSigidx);
		break;
	case 1:
		printf("refChooseSignal 1 = %d\n",refchSigidx);
		break;
	case 2:
		printf("refChooseSignal 2 = %d\n",refchSigidx);
		break;
	case 3:
		printf("refChooseSignal 3 = %d\n",refchSigidx);
		break;
	}


}




__global__ void GPU1(double* d_a,int row,int line,int dbn,int dbn_n)
{


}

__device__ void printMax(double* max,int dbn)
{
	printf("GPU打印\n");
    for(int i = 0; i < 4; ++i)
    {
        for(int j = 0; j < 2 * dbn; ++j)
        {
        	printf("%f\t",max[i * 2 * dbn + j]);
        }
        printf("\n");
    }
}

__device__ void printData(double* buffer,int row,int line)
{
    for(int i = 0; i < row; ++i)
    {
        for(int j = 0; j < line; ++j)
        {
        	printf("%d\t",buffer[i * line + j]);
        }
        printf("\n");
    }
}
// 这个函数用于开线程延拓线程开一组的就可以其他组的用line来区分
__global__ void decEx(double* dVectorUpSam,int line,int dbn,double* dVectorEx)
{
	int decExidx = threadIdx.x;//这个线程号最多只有一组的线程号

	for(int i = 0; i < 2 * dbn - 1; ++i)
	{
		dVectorEx[decExidx  * (line + 4 * dbn - 2) + i] 					 = dVectorUpSam[decExidx * line + 2 * dbn -2 - i];
		dVectorEx[decExidx  * (line + 4 * dbn - 2) + 2 * dbn + line - 1 + i] = dVectorUpSam[decExidx * line + line -1 - i];
	}
	for(int i = 0; i < line; ++i)
	{
		dVectorEx[decExidx  * (line + 4 * dbn - 2) + 2 * dbn - 1 + i] = dVectorUpSam[decExidx  * line + i];
	}
}

__device__ void decTransfromEx(double* d_data,double* dVectorUpSam,int line,int idx,int dbn,int decExLen,double* dVectorEx,int DBN_N)
{
	if(DBN_N == 0)
	{
		for(int i = 0; i < 2 * dbn - 1; ++i)
		{
			dVectorEx[i] 					  = d_data[idx * line + 2 * dbn -2 - i];
			dVectorEx[2 * dbn + line - 1 + i] = d_data[idx * line + line -1 - i];
		}
		for(int i = 0; i < line; ++i)
		{
			dVectorEx[2 * dbn - 1 + i] = d_data[idx * line + i];
		}
	}else{
		decEx<<<1,power(2,DBN_N)>>>(dVectorUpSam,line,dbn,dVectorEx);
	}
	return ;
}

__global__ void dataCON(double* dVectorEx,double* sdecMaxL,double* sdecMaxH,int dbn,int decCONLen,double* dVectorCON,int decCONidx,int decExLen,int DBN_N)
{
	int iidx = threadIdx.x;
	for(int i = 0; i < 2* dbn; ++i)
	{
		dVectorCON[2 * decCONidx * decCONLen + iidx] += dVectorEx[decCONidx * decExLen + iidx + i] * sdecMaxL[i];
		dVectorCON[(2 * decCONidx + 1) * decCONLen + iidx] += dVectorEx[decCONidx * decExLen + iidx + i] * sdecMaxH[i];
	}
}

__global__ void decTransfromCON(double* dVectorEx,double* sdecMaxL,double*sdecMaxH,int decCONLen,int dbn,double* dVectorCON,int decExLen,int DBN_N)
{
	int decCONidx = threadIdx.x;
	dataCON<<<1,decCONLen>>>(dVectorEx,sdecMaxL,sdecMaxH,dbn,decCONLen,dVectorCON,decCONidx,decExLen,DBN_N);
	__syncthreads();
}

__global__ void decUpSam(double* dVectorCON,double*dVectorUpSam,int decCONLen,int decUpSamLen,int DBN_N,int decUpidx)
{
	int iUpdx = threadIdx.x;
	dVectorUpSam[decUpSamLen * decUpidx + iUpdx] = dVectorCON[decCONLen * decUpidx + 2 * iUpdx + 1];
}

__global__ void decTransfromUpSam(double* dVectorCON,double*dVectorUpSam,int decCONLen,int decUpSamLen,int DBN_N)
{
	int decUpidx = threadIdx.x;
	decUpSam<<<1,decUpSamLen>>>(dVectorCON,dVectorUpSam,decCONLen,decUpSamLen,DBN_N,decUpidx);
	__syncthreads();
}

__device__ void dec(double* d_a,double* max,int row,int line,int dbn,int idx,int dbn_n,double*& refdata,int& refline)
{
	/*分解过程*/
	for(int DBN_N = 0; DBN_N < dbn_n; ++DBN_N)
	{
		int decExLen    = line + 4 * dbn - 2;//对称延拓长度
		int decCONLen   = line + 2 * dbn - 1;//分解卷积长度
		int decUpSamLen = (line + 2* dbn -1) / 2;//上采样长度

		double* dVectorEx 	  = new double[power(2,DBN_N) * decExLen]();
		double* dVectorCON    = new double[power(2,DBN_N + 1) * decCONLen]();		//  卷积和内存
		double* dVectorUpSam;													//  上采样内存

		decTransfromEx(d_a,dVectorUpSam,line,idx,dbn,decExLen,dVectorEx,DBN_N);//延拓

		printf("decExLen = %d\n",decExLen);
		for(int i = 0; i < power(2,DBN_N) * decExLen;++i)
		{
			printf("dVectorEx[%d] = %f\n",i,dVectorEx[i]);
		}
		printf("\n");

		double* sdecMaxL= new double[2 * dbn]();
		double* sdecMaxH= new double[2 * dbn]();

		for(int i = 0; i < 2 * dbn; ++i)
		{
			sdecMaxL[i] = max[0 * 2 * dbn + i];
			sdecMaxH[i] = max[1 * 2 * dbn + i];
		}
		decTransfromCON<<<1,power(2,DBN_N)>>>(dVectorEx,sdecMaxL,sdecMaxH,decCONLen,dbn,dVectorCON,decExLen,DBN_N);//卷积
		__syncthreads();
		hipFree(dVectorEx);

		printf("decCONLen = %d\n",decCONLen);
		for(int i = 0; i < power(2,DBN_N + 1) * decCONLen;++i)
		{
			printf("dVectorCON[%d] = %f\n",i,dVectorCON[i]);
		}
		dVectorUpSam = new double[power(2,DBN_N + 1) * decUpSamLen]();
		decTransfromUpSam<<<1,power(2,DBN_N + 1)>>>(dVectorCON,dVectorUpSam,decCONLen,decUpSamLen,DBN_N);
		__syncthreads();
		hipFree(dVectorCON);
		hipFree(sdecMaxL);
		hipFree(sdecMaxH);
		printf("decUpSamLen = %d\n",decUpSamLen);
		for(int i = 0; i < power(2,DBN_N + 1) * decUpSamLen;++i)
		{
			printf("dVectorUpSam[%d] = %f\n",i,dVectorUpSam[i]);
		}
		//更新系数,进行下一层分解
		line = decUpSamLen;
		if(DBN_N == dbn_n - 1)//重构信号长度
		{
			refline = line;
			refdata = new double[power(2,dbn_n)*refline];
			refdata = dVectorUpSam;
			hipFree(dVectorUpSam);
		}
	}
}


