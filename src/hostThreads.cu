#include "head.h"

void threadHostToDevice0(std::vector<double>& Hvec, std::vector<double>& maxVector,int row, int line, int dbn,int dbn_n)//传引用
{
    std::cout << "CPU线程操作" << std::endl;
    int decDataLen = line;
    int decExLen = 0;
    int decCONLen = 0;
    int decUpSamLen = 0;
    for(int i = 0; i < dbn_n; ++i)
    {
    	decExLen  = decDataLen + 4 * dbn - 2;//对称延拓长度
    	decCONLen = decDataLen + 2 * dbn - 1;//分解卷积长度
		int decUpSamLen = (decDataLen + 2* dbn -1) / 2;//上采样长度
		decDataLen = decUpSamLen;
    }
    printf("decExLen = %d\n",decExLen);
    printf("decCONLen = %d\n",decCONLen);
    printf("decDataLen = %d\n",decDataLen);
    hipDeviceSetLimit(hipLimitMallocHeapSize,1024 * 1024 * 1024);
	hipSetDevice(0);																//启动GPU0
    double* DeviceDecDataFinsh;														//分解结果的数据
    double* HostDecDataFinsh = new double[decDataLen * HD_power(dbn_n) * row]();   //分解结果的数据
	double* d_a;																	//申明显存地址
	double* max_data;																//申请矩阵地址
	double* maxBuffer = new double[8 * dbn]();
	double* buffer = new double[row * line](); 								//申请数组内存
	double* dVectorEx;			//分解延拓
	double* dVectorCON;			//分解卷积
	double* dVectorUpSam;		//分解上采样

	if(!Hvec.empty())									//把vector的数组转换到普通数组
	{
		memcpy(buffer,&Hvec[0],Hvec.size() * sizeof(double));
	}else
	{
		std::cout << "数据初始化失败"<< std::endl;
		exit(-1);
	}

	if(!maxVector.empty())									//把vector的数组转换到普通数组
	{
		memcpy(maxBuffer,&maxVector[0],maxVector.size() * sizeof(double));
	}else
	{
		std::cout << "矩阵初始化失败"<< std::endl;
		exit(-1);
	}
	//测试矩阵
	for(int i = 0; i < 4; ++i)
	{
		for(int j = 0; j < 2*dbn;++j)
		{
			maxBuffer[i * 2 * dbn + j] = 1;
		}
	}
	hipMalloc((void**)&dVectorEx,sizeof(double) * decExLen * HD_power(dbn_n) * row); //分解延拓内存
	hipMalloc((void**)&dVectorCON,sizeof(double) * decCONLen * HD_power(dbn_n + 1) * row); //分解卷积内存
	hipMalloc((void**)&DeviceDecDataFinsh,sizeof(double) * decDataLen * HD_power(dbn_n + 1) * row); //分解上采样内存,分解结果数据

//这个函数是用来清空申请的显存的.
//	hipMemset(dVectorEx,0,sizeof(double) * decExLen * HD_power(dbn_n) * row);
//	hipMemset(dVectorCON,0,sizeof(double) * decCONLen * HD_power(dbn_n+1) * row);
//	hipMemset(DeviceDecDataFinsh,0,sizeof(double) * decDataLen * HD_power(dbn_n+1) * row);
    hipMalloc((void**)&max_data,sizeof(double) * 8 * dbn); //申请矩阵内存
	hipMalloc((void**)&d_a,sizeof(double) * row * line);	//申请显存大小
	hipMemcpy(d_a,buffer,sizeof(double) * row * line,hipMemcpyHostToDevice);//内存数据导入到显存
	hipMemcpy(max_data,maxBuffer,sizeof(double) * 8 * dbn,hipMemcpyHostToDevice);

	GPU0<<<1,row>>>(DeviceDecDataFinsh,dVectorEx,dVectorCON,d_a,max_data,row,line,dbn,dbn_n,decDataLen);					//启动核函数
	hipMemcpy(HostDecDataFinsh,DeviceDecDataFinsh,sizeof(double) * decDataLen * HD_power(dbn_n) * row,hipMemcpyDeviceToHost);
	hipDeviceReset();

//	for(int i = 0; i < decDataLen * H_power(2,dbn_n); ++i)
//	{
//		printf("HostDecDataFinsh[%d] = %f\n",i,HostDecDataFinsh[i]);
//	}
	mkdir_file(1,1);
	for(int k = 0; k < row; ++k)
	{
	    std::ofstream WD(mkdir_txt(1,1,k + 1));
	    for(uint i = 0; i < HD_power(dbn_n); ++i)
	    {
	        for(uint j = 0; j < decDataLen; ++j)
	        {
	            WD << std::setprecision(16) << HostDecDataFinsh[k * HD_power(dbn_n) * decDataLen + decDataLen * i + j] << " ";
	        }
	        WD << std::endl;
	    }
	}

	hipFree(d_a);									//释放内存
	delete[] buffer;
	delete[] maxBuffer;
	hipDeviceReset();								//重置GPU0


}

void threadHostToDevice1()
{

}

void threadDeviceToHost0()
{

}

void threadDeviceToHost1()
{

}
