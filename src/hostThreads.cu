#include "head.h"


void threadHostToDevice0(const std::vector<double>& Hvec,const int& row,const int& line,const int& dbn_n)//传引用
{
	hipSetDevice(0);								//启动GPU0
	double *d_a;									//申明显存地址
	double *buffer = new double[row*line];			//申请数组内存
	const std::vector<double> A = Hvec;
	if(!A.empty())									//把vector的数组转换到普通数组
	{
		memcpy(buffer,&A[0],A.size() * sizeof(double));
	}
	hipMalloc((void**)&d_a,sizeof(double)*row*line);	//申请显存大小
	hipMemcpy(d_a,buffer,sizeof(double)*row*line,hipMemcpyHostToDevice);//内存数据导入到显存

	GPU0<<<1,1>>>(d_a,row,line,dbn_n);					//启动核函数

	hipFree(d_a);									//释放内存

	hipDeviceReset();								//重置GPU0


}

void threadHostToDevice1()
{

}

void threadDeviceToHost0()
{

}

void threadDeviceToHost1()
{

}
