#include "head.h"

void threadHostToDevice0(std::vector<double>& Hvec, std::vector<double>& maxVector,int row, int line, int dbn,int dbn_n)//传引用
{
    std::cout << "CPU线程操作" << std::endl;
	hipSetDevice(0);								//启动GPU0
	double* d_a;									//申明显存地址
	double* max_data;								//申请矩阵地址
	double* maxBuffer = new double[8 * dbn];
	double* buffer = new double[row * line]; 		//申请数组内存
	thrust::host_vector<int> h_line;
	thrust::device_vector<int> d_line;

	if(!Hvec.empty())									//把vector的数组转换到普通数组
	{
		memcpy(buffer,&Hvec[0],Hvec.size() * sizeof(double));
	}else
	{
		std::cout << "数据初始化失败"<< std::endl;
		exit(-1);
	}
    for(int i = 0; i < row; ++i)
    {
        for(int j = 0; j < line; ++j)
        {
            std::cout << buffer[i * line + j] << " ";
        }
        std::cout<< std::endl;
    }

	if(!maxVector.empty())									//把vector的数组转换到普通数组
	{
//		memcpy(maxBuffer,&maxVector[0],maxVector.size() * sizeof(double));
		//测试矩阵
		for(int i = 0; i < 4;i++)
		{
			for(int j = 0; j < 2 * dbn; ++j)
			{
				maxBuffer[i * 2 * dbn + j] = i * 2 * dbn + j + 1;
			}
		}
	}else
	{
		std::cout << "矩阵初始化失败"<< std::endl;
		exit(-1);
	}
    for(int i = 0; i < 4; ++i)
    {
        for(int j = 0; j < 2 * dbn; ++j)
        {
            std::cout << maxBuffer[i * 2 * dbn + j] << " ";
        }
        std::cout<< std::endl;
    }
//    hipMalloc(&d_line,sizeof(int));
    hipMalloc((void**)&max_data,sizeof(double) * 8 * dbn); //申请矩阵内存
	hipMalloc((void**)&d_a,sizeof(double) * row * line);	//申请显存大小
	hipMemcpy(d_a,buffer,sizeof(double) * row * line,hipMemcpyHostToDevice);//内存数据导入到显存
	hipMemcpy(max_data,maxBuffer,sizeof(double) * 8 * dbn,hipMemcpyHostToDevice);

	GPU0<<<1,4>>>(d_a,max_data,row,line,dbn,dbn_n);					//启动核函数

	h_line = d_line;


	hipFree(d_a);									//释放内存
	delete[] buffer;
	delete[] maxBuffer;
	hipDeviceReset();								//重置GPU0


}

void threadHostToDevice1()
{

}

void threadDeviceToHost0()
{

}

void threadDeviceToHost1()
{

}
