#include "head.h"

void threadHostToDevice0(std::vector<double>& Hvec, std::vector<double>& maxVector,int row, int line, int dbn,int dbn_n)//传引用
{
    std::cout << "CPU线程操作" << std::endl;
    int decDataLen = line;
    for(int i = 0; i < dbn_n; ++i)
    {
		int decUpSamLen = (decDataLen + 2* dbn -1) / 2;//上采样长度
		decDataLen = decUpSamLen;
    }
	hipSetDevice(0);																//启动GPU0
    double* DeviceDecDataFinsh;														//分解结果的数据
    double* HostDecDataFinsh = new double[decDataLen * H_power(2,dbn_n) * row]();   //分解结果的数据
	double* d_a;																	//申明显存地址
	double* max_data;																//申请矩阵地址
	double* maxBuffer = new double[8 * dbn]();
	double* buffer = new double[row * line](); 								//申请数组内存

	if(!Hvec.empty())									//把vector的数组转换到普通数组
	{
		memcpy(buffer,&Hvec[0],Hvec.size() * sizeof(double));
	}else
	{
		std::cout << "数据初始化失败"<< std::endl;
		exit(-1);
	}
	std::ofstream WD("../测试数据/1.txt");
    for(int i = 0; i < row; ++i)
    {
        for(int j = 0; j < line; ++j)
        {
            WD << std::setprecision(16) << buffer[i * line + j]<< " ";
        }
        WD << std::endl;
    }

	if(!maxVector.empty())									//把vector的数组转换到普通数组
	{
		memcpy(maxBuffer,&maxVector[0],maxVector.size() * sizeof(double));
	}else
	{
		std::cout << "矩阵初始化失败"<< std::endl;
		exit(-1);
	}

    hipMalloc((void**)&DeviceDecDataFinsh,sizeof(double) * decDataLen * H_power(2,dbn_n) * row); //分解结果数据
    hipMalloc((void**)&max_data,sizeof(double) * 8 * dbn); //申请矩阵内存
	hipMalloc((void**)&d_a,sizeof(double) * row * line);	//申请显存大小
	hipMemcpy(d_a,buffer,sizeof(double) * row * line,hipMemcpyHostToDevice);//内存数据导入到显存
	hipMemcpy(max_data,maxBuffer,sizeof(double) * 8 * dbn,hipMemcpyHostToDevice);

	GPU0<<<1,row>>>(DeviceDecDataFinsh,d_a,max_data,row,line,dbn,dbn_n,decDataLen);					//启动核函数

	hipMemcpy(HostDecDataFinsh,DeviceDecDataFinsh,sizeof(double) * decDataLen * H_power(2,dbn_n) * row,hipMemcpyDeviceToHost);

//	for(int i = 0; i < decDataLen * H_power(2,dbn_n); ++i)
//	{
//		printf("HostDecDataFinsh[%d] = %f\n",i,HostDecDataFinsh[i]);
//	}
	mkdir_file(1,1);
	for(int k = 0; k < row; ++k)
	{
	    std::ofstream WD(mkdir_txt(1,1,k + 1));
	    for(uint i = 0; i < H_power(2,dbn_n); ++i)
	    {
	        for(uint j = 0; j < decDataLen; ++j)
	        {
	            WD << std::setprecision(16) << HostDecDataFinsh[k * H_power(2,dbn_n) * decDataLen + decDataLen * i + j] << " ";
	        }
	        WD << std::endl;
	    }
	}

	hipFree(d_a);									//释放内存
	delete[] buffer;
	delete[] maxBuffer;
	hipDeviceReset();								//重置GPU0


}

void threadHostToDevice1()
{

}

void threadDeviceToHost0()
{

}

void threadDeviceToHost1()
{

}
