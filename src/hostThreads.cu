#include "head.h"


void threadHostToDevice0(std::vector<double>& Hvec, int row, int line, int dbn,int dbn_n)//传引用
{
    std::cout << "CPU线程操作" << std::endl;
	hipSetDevice(0);								//启动GPU0
	double *d_a;									//申明显存地址
	double *buffer = new double[row*line];			//申请数组内存
	const std::vector<double> A = Hvec;
	if(!A.empty())									//把vector的数组转换到普通数组
	{
		memcpy(buffer,&A[0],A.size() * sizeof(double));
	}
    for(std::vector<double>::size_type i = 0; i < row; ++i)
    {
        for(std::vector<double>::size_type j = 0; j < line; ++j)
        {
            std::cout << buffer[i * line + j] << " ";
        }
        std::cout<< std::endl;
    }

	hipMalloc((void**)&d_a,sizeof(double)*row*line);	//申请显存大小
	hipMemcpy(d_a,buffer,sizeof(double)*row*line,hipMemcpyHostToDevice);//内存数据导入到显存


	GPU0<<<1,4>>>(d_a,row,line,dbn,dbn_n);					//启动核函数

	hipFree(d_a);									//释放内存

	hipDeviceReset();								//重置GPU0


}

void threadHostToDevice1()
{

}

void threadDeviceToHost0()
{

}

void threadDeviceToHost1()
{

}
