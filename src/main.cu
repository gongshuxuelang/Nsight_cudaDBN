#include "hip/hip_runtime.h"
#include "head.h"
//字母说明，dec为分解，signal为信号，Len为长度，filter为滤波器，
//EX为延拓，CON为卷积，DSam为下采样，D为下，sam为采样。L为低频信号，H为高频信号
//ref为重构，
void funic(std::vector<double>& q)
{
	std::vector<double> a = q;
}
int main()
{
	//初始化小波分解所需要的系数，第一个参数为DBn小波，第二个系数是信号的行数，第三个系数是信号的列数
    DWT dwt(4, 3, 4, 5, 0, 1, 1, 32); //参数说明，，第一参数是dbn小波，第二个参数是分解n层，第三个参数是有
    std::vector<double> sp;
    std::cout << "测试点" << std::endl;
    Matrix max(dwt.getDWT_DBN());
    max.matrix();					//创建系数矩阵
    std::cout << "打印系数矩阵" << std::endl;
//    max.Print_matrix();
    sp = max.getMatrix();

    std::cout << "系数矩阵返回值" << std::endl;
    for(int i = 0; i < 4; ++i)
    {
    	for(int j = 0; j < 2 * dwt.getDWT_DBN();++j)
    	{
    		std::cout << "sp["<< i << "][" << j << "] = " << sp[i * 2 * dwt.getDWT_DBN()  + j] <<"  ";
    	}
    	std::cout << std::endl;
    }

    //测试读文件
    std::vector<double> rdt;
    rwDate_dec rdc(dwt.getDWT_DBN(),dwt.getDWT_data_row(),dwt.getDWT_data_line(),dwt.getDWT_file_m(),dwt.getDWT_file_n(),dwt.getDWT_mode());
    rdc.ReadDate();
//    rdc.Print_rwDate();
    //测试返回值
    rdt = rdc.getRaw_Data();
    std::cout << "rdt.size() = " << rdt.size() << std::endl;
    for(std::vector<double>::size_type i = 0; i < dwt.getDWT_data_row(); ++i)
    {
        for(std::vector<double>::size_type j = 0; j < dwt.getDWT_data_line(); ++j)
        {
            std::cout << rdt[i * dwt.getDWT_data_line() + j] << " ";
        }
        std::cout<< std::endl;
    }

    funic(rdt);

    boost::thread t_HtoD0(threadHostToDevice0,boost::cref(rdt));
    t_HtoD0.join();


	return 0;
}
