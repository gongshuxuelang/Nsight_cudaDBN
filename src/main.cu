#include "hip/hip_runtime.h"
#include "head.h"
//字母说明，dec为分解，signal为信号，Len为长度，filter为滤波器，
//EX为延拓，CON为卷积，DSam为下采样，D为下，sam为采样。L为低频信号，H为高频信号
//ref为重构，

int main()
{
	//初始化小波分解所需要的系数，第一个参数为DBn小波，第二个系数是信号的行数，第三个系数是信号的列数
    DWT dwt(2, 3, 4, 8, 0, 1, 1, 32); //参数说明，，第一参数是dbn小波，第二个参数是分解n层，第三个参数是有
    std::vector<double> maxVector;//接收矩阵函数
    std::cout << "测试点" << std::endl;
    Matrix max(dwt.getDWT_DBN());
    max.matrix();					//创建系数矩阵
    maxVector = max.getMatrix();
    //测试读文件
    std::vector<double> rdt;
    rwDate_dec rdc(dwt.getDWT_DBN(),dwt.getDWT_data_row(),dwt.getDWT_data_line(),dwt.getDWT_file_m(),dwt.getDWT_file_n(),dwt.getDWT_mode());
    rdc.ReadDate();
    //rdc.Print_rwDate();
    //测试返回值
    rdt = rdc.getRaw_Data();

    boost::thread t_HtoD0(threadHostToDevice0,boost::ref(rdt),boost::ref(maxVector),dwt.getDWT_data_row(),dwt.getDWT_data_line(),dwt.getDWT_DBN(),dwt.getDWT_DBN_N());
    t_HtoD0.join();

    std::cout << "运行完毕" << std::endl;

	return 0;
}
