#include "hip/hip_runtime.h"
#include "head.h"

//字母说明，dec为分解，signal为信号，Len为长度，filter为滤波器，
 //EX为延拓，CON为卷积，DSam为下采样，D为下，sam为采样。L为低频信号，H为高频信号
 //ref为重构，
__global__ void gpu()
{
    printf("hello gpu\n");
}

int main()
{
      
    //初始化小波分解所需要的系数，第一个参数为DBn小波，第二个系数是信号的行数，第三个系数是信号的列数
    DWT dwt(4,9,32,7681,0,32,40,32);//参数说明，，第一参数是dbn小波，第二个参数是分解n层，第三个参数是有
    Matrix max(4);
    max.Matrix_Init();
    max.creatMatrix();
    max.Print_matrix();
    hipSetDevice(0);
    gpu<<<1,20>>>();
    hipDeviceReset();
     std::cout << "读文件： " << std::endl;
    rwDate_dec rwdDec(dwt.getDWT_DBN(),dwt.getDWT_data_row(),dwt.getDWT_data_line(),1,1,dwt.getDWT_mode());
    //测试点
    std::cout << "打印数据" << std::endl;
    rwdDec.ReadDate();
    rwdDec.Print_rwDate();

    return 0;
}
